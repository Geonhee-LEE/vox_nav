// Copyright (c) 2020 Fetullah Atas, Norwegian University of Life Sciences
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <rclcpp/rclcpp.hpp>
#include <sensor_msgs/msg/point_cloud2.hpp>
#include <sensor_msgs/msg/imu.hpp>
#include <nav_msgs/msg/odometry.hpp>
#include <message_filters/synchronizer.h>
#include <message_filters/subscriber.h>
#include <message_filters/sync_policies/approximate_time.h>
#include <tf2_geometry_msgs/tf2_geometry_msgs.h>

#include <pcl/conversions.h>
#include <pcl/point_cloud.h>
#include <pcl/point_types.h>
#include <pcl/common/io.h>
#include <pcl/PCLPointCloud2.h>
#include <pcl_conversions/pcl_conversions.h>
#include <tf2_eigen/tf2_eigen.h>
#include <pcl/common/common.h>
#include <pcl/filters/extract_indices.h>
#include <pcl/filters/crop_box.h>
#include <pcl_ros/transforms.hpp>

#include "cupoch/cupoch.h"
#include "vox_nav_utilities/pcl_helpers.hpp"
#include "vox_nav_utilities/tf_helpers.hpp"

#include <queue>
#include <ostream>
#include <iostream>
#include <sstream>
#include <iomanip>
#include <fstream>

namespace vox_nav_utilities
{
  /**
   * @brief Given topic of cloud, odometry and imu,
   * writes a dataset in KITTI format.
   *
   */
  class Ros2BagKittiBin : public rclcpp::Node
  {

  public:
    /**
     * @brief Construct a new Ros 2 Bag Kitti Bin object
     *
     */
    Ros2BagKittiBin();

    /**
     * @brief Destroy the Ros 2 Bag Kitti Bin object
     *
     */
    ~Ros2BagKittiBin();

    /**
     * @brief message filter to sync multiple data topics
     *
     */
    typedef message_filters::sync_policies::ApproximateTime<
        sensor_msgs::msg::PointCloud2,
        nav_msgs::msg::Odometry,
        sensor_msgs::msg::Imu>
      CloudOdomApprxTimeSyncPolicy;
    typedef message_filters::Synchronizer<CloudOdomApprxTimeSyncPolicy>
      CloudOdomApprxTimeSyncer;


    /**
     * @brief recieves all these messages and writes them to a directory
     *
     * @param cloud
     * @param odom
     * @param imu
     */
    void cloudOdomCallback(
      const sensor_msgs::msg::PointCloud2::ConstSharedPtr & cloud,
      const nav_msgs::msg::Odometry::ConstSharedPtr & odom,
      const sensor_msgs::msg::Imu::ConstSharedPtr & imu);

    /**
     * @brief write a PCL type cloud to file in binary format
     *
     * @param in_cloud
     * @param out_file
     */
    void pcd2bin(pcl::PointCloud<pcl::PointXYZI>::Ptr in_cloud, std::string & out_file);

  private:
    message_filters::Subscriber<sensor_msgs::msg::PointCloud2> cloud_subscriber_;
    message_filters::Subscriber<nav_msgs::msg::Odometry> odom_subscriber_;
    message_filters::Subscriber<sensor_msgs::msg::Imu> imu_subscriber_;
    std::shared_ptr<CloudOdomApprxTimeSyncer> cloud_odom_data_approx_time_syncher_;

    int sequence_horizon_;
    double dt_;
    double sensor_height_;
    std::string save_dir_;

    bool recieved_first_;
    int file_index_;

    std::string poses_;
    std::ofstream poses_infile_;

    std::vector<std::tuple<
        sensor_msgs::msg::PointCloud2::SharedPtr,
        nav_msgs::msg::Odometry::SharedPtr,
        sensor_msgs::msg::Imu::SharedPtr>>
    cloud_odom_vector_;

    rclcpp::Time last_recieved_msg_stamp_;
    rclcpp::Time stamp_;

    double roll_, pitch_, yaw_;
    double x_, y_, z_;
  };

  Ros2BagKittiBin::Ros2BagKittiBin()
  : Node("dynamic_points_node"),
    recieved_first_(false),
    file_index_(0)
  {
    cloud_subscriber_.subscribe(this, "points", rmw_qos_profile_sensor_data);
    odom_subscriber_.subscribe(this, "odom", rmw_qos_profile_sensor_data);
    imu_subscriber_.subscribe(this, "imu", rmw_qos_profile_sensor_data);

    declare_parameter("sequence_horizon", 0);
    get_parameter("sequence_horizon", sequence_horizon_);
    declare_parameter("dt", 0.0);
    get_parameter("dt", dt_);
    declare_parameter("sensor_height", 0.0);
    get_parameter("sensor_height", sensor_height_);
    declare_parameter("save_dir", "/path/to/save_dir");
    get_parameter("save_dir", save_dir_);

    cloud_odom_data_approx_time_syncher_.reset(
      new CloudOdomApprxTimeSyncer(
        CloudOdomApprxTimeSyncPolicy(1000),
        cloud_subscriber_,
        odom_subscriber_,
        imu_subscriber_));

    cloud_odom_data_approx_time_syncher_->registerCallback(
      std::bind(
        &Ros2BagKittiBin::cloudOdomCallback, this,
        std::placeholders::_1,
        std::placeholders::_2,
        std::placeholders::_3));

    poses_ = save_dir_ + std::string("/poses.txt");
    poses_infile_.open(poses_.c_str());
  }

  Ros2BagKittiBin::~Ros2BagKittiBin()
  {
    poses_infile_.close();
  }

  void Ros2BagKittiBin::cloudOdomCallback(
    const sensor_msgs::msg::PointCloud2::ConstSharedPtr & cloud,
    const nav_msgs::msg::Odometry::ConstSharedPtr & odom,
    const sensor_msgs::msg::Imu::ConstSharedPtr & imu)
  {

    stamp_ = cloud->header.stamp;

    pcl::PointCloud<pcl::PointXYZI>::Ptr curr_pcl(new pcl::PointCloud<pcl::PointXYZI>());

    pcl::PCLPointCloud2 pcl_pc;
    pcl_conversions::toPCL(*cloud, pcl_pc);
    pcl::fromPCLPointCloud2(pcl_pc, *curr_pcl);

    tf2::Transform T;
    T.setOrigin(tf2::Vector3(0, 0, sensor_height_));
    T.setRotation(tf2::Quaternion::getIdentity());
    pcl_ros::transformPointCloud(*curr_pcl, *curr_pcl, T);

    std::stringstream buffer;
    buffer << std::setfill('0') << std::setw(6) << file_index_;

    // write the first arrived message set regardless
    if (!recieved_first_) {
      recieved_first_ = true;

      last_recieved_msg_stamp_ = cloud->header.stamp;

      tf2::Transform T;
      x_ = odom->pose.pose.position.x;
      y_ = odom->pose.pose.position.y;
      z_ = odom->pose.pose.position.z;

      vox_nav_utilities::getRPYfromMsgQuaternion(
        odom->pose.pose.orientation,
        roll_,
        pitch_,
        yaw_);

      auto trans = Eigen::Vector3f(
        odom->pose.pose.position.x /*- x_*/,
        odom->pose.pose.position.y /*- y_*/,
        odom->pose.pose.position.z /*- z_*/);

      double yaw_latest, pitch_latest, roll_latest;

      vox_nav_utilities::getRPYfromMsgQuaternion(
        odom->pose.pose.orientation,
        roll_latest,
        pitch_latest,
        yaw_latest);

      auto rot = cupoch::geometry::GetRotationMatrixFromXYZ(
        Eigen::Vector3f(roll_latest, pitch_latest, yaw_latest));

      Eigen::Matrix4f pose = Eigen::Matrix4f::Identity();

      pose.block<3, 3>(0, 0) = rot;
      pose.block<3, 1>(0, 3) = trans;

      for (size_t i = 0; i < 3; i++) {
        for (size_t j = 0; j < 4; j++) {
          poses_infile_ << pose(i, j);
          poses_infile_ << " ";
        }
      }
      poses_infile_ << "\n";

      std::string out_file = save_dir_ + "/velodyne/" + buffer.str() + ".bin";
      pcd2bin(curr_pcl, out_file);

      RCLCPP_INFO(get_logger(), " Writing a cloud with %d points", curr_pcl->points.size());

      file_index_++;
    }

    // Let some dt_ amount of time to pass in between frames
    if ((stamp_ - last_recieved_msg_stamp_).seconds() > dt_) {
      last_recieved_msg_stamp_ = cloud->header.stamp;

      auto trans = Eigen::Vector3f(
        odom->pose.pose.position.x /*- x_*/,
        odom->pose.pose.position.y /*- y_*/,
        odom->pose.pose.position.z /*- z_*/);

      double yaw_latest, pitch_latest, roll_latest;

      vox_nav_utilities::getRPYfromMsgQuaternion(
        odom->pose.pose.orientation,
        roll_latest,
        pitch_latest,
        yaw_latest);

      auto rot = cupoch::geometry::GetRotationMatrixFromXYZ(
        Eigen::Vector3f(roll_latest, pitch_latest, yaw_latest));

      Eigen::Matrix4f pose = Eigen::Matrix4f::Identity();

      pose.block<3, 3>(0, 0) = rot;
      pose.block<3, 1>(0, 3) = trans;

      for (size_t i = 0; i < 3; i++) {
        for (size_t j = 0; j < 4; j++) {
          poses_infile_ << pose(i, j);
          poses_infile_ << " ";
        }
      }
      poses_infile_ << "\n";

      std::string out_file = save_dir_ + "/velodyne/" + buffer.str() + ".bin";
      pcd2bin(curr_pcl, out_file);

      RCLCPP_INFO(get_logger(), " Writing a cloud with %d points", curr_pcl->points.size());

      file_index_++;
    }
  }

  // Transform PCD 2 BIN
  void Ros2BagKittiBin::pcd2bin(
    pcl::PointCloud<pcl::PointXYZI>::Ptr in_cloud,
    std::string & out_file)
  {
    using namespace std;

    std::ofstream bin_file(out_file.c_str(), std::ios::out | std::ios::binary | std::ios::app);
    if (!bin_file.good()) {
      RCLCPP_INFO(get_logger(), "Couldn't open %s", out_file.c_str());
    }

    for (size_t i = 0; i < in_cloud->points.size(); ++i) {
      bin_file.write((char *)&in_cloud->points[i].x, 3 * sizeof(float));
      bin_file.write((char *)&in_cloud->points[i].intensity, sizeof(float));
    }

    bin_file.close();
  }

} // namespace

int main(int argc, char const * argv[])
{
  rclcpp::init(argc, argv);
  cupoch::utility::InitializeAllocator();
  auto node = std::make_shared<vox_nav_utilities::Ros2BagKittiBin>();
  rclcpp::spin(node);
  rclcpp::shutdown();
  return 0;
}
