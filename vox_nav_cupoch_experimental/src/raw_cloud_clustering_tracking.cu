#include "hip/hip_runtime.h"
// Copyright (c) 2020 Fetullah Atas, Norwegian University of Life Sciences
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

/*
DISCLAIMER: some parts of code has been taken from; https://github.com/appinho/SARosPerceptionKitti
Credits to author: Simon Appel, https://github.com/appinho
*/

#include "vox_nav_cupoch_experimental/raw_cloud_clustering_tracking.hpp"

using namespace vox_nav_cupoch_experimental;

RawCloudClusteringTracking::RawCloudClusteringTracking()
    : Node("cloud_clustering_rclcpp_node")
{
    cloud_subscriber_.subscribe(this, "points", rmw_qos_profile_sensor_data);
    poses_subscriber_.subscribe(this, "poses", rmw_qos_profile_sensor_data);

    cloud_poses_data_approx_time_syncher_.reset(
        new CloudOdomApprxTimeSyncer(
            CloudOdomApprxTimeSyncPolicy(500),
            cloud_subscriber_,
            poses_subscriber_));

    cloud_poses_data_approx_time_syncher_->registerCallback(
        std::bind(
            &RawCloudClusteringTracking::cloudOdomCallback, this,
            std::placeholders::_1,
            std::placeholders::_2));

    cloud_clusters_pub_ = this->create_publisher<sensor_msgs::msg::PointCloud2>(
        "detection/clusters", rclcpp::SystemDefaultsQoS());

    tracking_markers_pub_ = this->create_publisher<visualization_msgs::msg::MarkerArray>(
        "tracking/objects", rclcpp::SystemDefaultsQoS());

    // Define parameters
    declare_parameter("data_association.ped.dist.position", params_.da_ped_dist_pos);
    declare_parameter("data_association.ped.dist.form", params_.da_ped_dist_form);
    declare_parameter("data_association.car.dist.position", params_.da_car_dist_pos);
    declare_parameter("data_association.car.dist.form", params_.da_car_dist_form);
    declare_parameter("tracking.dim.z", params_.tra_dim_z);
    declare_parameter("tracking.dim.x", params_.tra_dim_x);
    declare_parameter("tracking.dim.x_aug", params_.tra_dim_x_aug);
    declare_parameter("tracking.std.lidar.x", params_.tra_std_lidar_x);
    declare_parameter("tracking.std.lidar.y", params_.tra_std_lidar_y);
    declare_parameter("tracking.std.acc", params_.tra_std_acc);
    declare_parameter("tracking.std.yaw_rate", params_.tra_std_yaw_rate);
    declare_parameter("tracking.lambda", params_.tra_lambda);
    declare_parameter("tracking.aging.bad", params_.tra_aging_bad);
    declare_parameter("tracking.occlusion_factor", params_.tra_occ_factor);
    declare_parameter("tracking.min_dist_between_tracks", params_.tra_min_dist_between_tracks);
    declare_parameter("track.P_init.x", params_.p_init_x);
    declare_parameter("track.P_init.y", params_.p_init_y);
    declare_parameter("track.P_init.v", params_.p_init_v);
    declare_parameter("track.P_init.yaw", params_.p_init_yaw);
    declare_parameter("track.P_init.yaw_rate", params_.p_init_yaw_rate);
    declare_parameter("clustering.x_bound", clustering_params_.x_bound);
    declare_parameter("clustering.y_bound", clustering_params_.y_bound);
    declare_parameter("clustering.z_bound", clustering_params_.z_bound);
    declare_parameter("clustering.downsample_voxel_size", clustering_params_.downsample_voxel_size);
    declare_parameter("clustering.remove_ground_plane_thres", clustering_params_.remove_ground_plane_thres);
    declare_parameter("clustering.clustering_min_points", clustering_params_.clustering_min_points);
    declare_parameter("clustering.clustering_max_points", clustering_params_.clustering_max_points);
    declare_parameter("clustering.clustering_max_step_size", clustering_params_.clustering_max_step_size);
    declare_parameter("clustering.sacle_up_objects", clustering_params_.sacle_up_objects);

    get_parameter("data_association.ped.dist.position", params_.da_ped_dist_pos);
    get_parameter("data_association.ped.dist.form", params_.da_ped_dist_form);
    get_parameter("data_association.car.dist.position", params_.da_car_dist_pos);
    get_parameter("data_association.car.dist.form", params_.da_car_dist_form);
    get_parameter("tracking.dim.z", params_.tra_dim_z);
    get_parameter("tracking.dim.x", params_.tra_dim_x);
    get_parameter("tracking.dim.x_aug", params_.tra_dim_x_aug);
    get_parameter("tracking.std.lidar.x", params_.tra_std_lidar_x);
    get_parameter("tracking.std.lidar.y", params_.tra_std_lidar_y);
    get_parameter("tracking.std.acc", params_.tra_std_acc);
    get_parameter("tracking.std.yaw_rate", params_.tra_std_yaw_rate);
    get_parameter("tracking.lambda", params_.tra_lambda);
    get_parameter("tracking.aging.bad", params_.tra_aging_bad);
    get_parameter("tracking.occlusion_factor", params_.tra_occ_factor);
    get_parameter("tracking.min_dist_between_tracks", params_.tra_min_dist_between_tracks);
    get_parameter("track.P_init.x", params_.p_init_x);
    get_parameter("track.P_init.y", params_.p_init_y);
    get_parameter("track.P_init.v", params_.p_init_v);
    get_parameter("track.P_init.yaw", params_.p_init_yaw);
    get_parameter("track.P_init.yaw_rate", params_.p_init_yaw_rate);
    get_parameter("clustering.x_bound", clustering_params_.x_bound);
    get_parameter("clustering.y_bound", clustering_params_.y_bound);
    get_parameter("clustering.z_bound", clustering_params_.z_bound);
    get_parameter("clustering.downsample_voxel_size", clustering_params_.downsample_voxel_size);
    get_parameter("clustering.remove_ground_plane_thres", clustering_params_.remove_ground_plane_thres);
    get_parameter("clustering.clustering_min_points", clustering_params_.clustering_min_points);
    get_parameter("clustering.clustering_max_points", clustering_params_.clustering_max_points);
    get_parameter("clustering.clustering_max_step_size", clustering_params_.clustering_max_step_size);
    get_parameter("clustering.sacle_up_objects", clustering_params_.sacle_up_objects);

    // Print parameters
    RCLCPP_INFO_STREAM(get_logger(), "da_ped_dist_pos " << params_.da_ped_dist_pos);
    RCLCPP_INFO_STREAM(get_logger(), "da_ped_dist_form " << params_.da_ped_dist_form);
    RCLCPP_INFO_STREAM(get_logger(), "da_car_dist_pos " << params_.da_car_dist_pos);
    RCLCPP_INFO_STREAM(get_logger(), "da_car_dist_form " << params_.da_car_dist_form);
    RCLCPP_INFO_STREAM(get_logger(), "tra_dim_z " << params_.tra_dim_z);
    RCLCPP_INFO_STREAM(get_logger(), "tra_dim_x " << params_.tra_dim_x);
    RCLCPP_INFO_STREAM(get_logger(), "tra_dim_x_aug " << params_.tra_dim_x_aug);
    RCLCPP_INFO_STREAM(get_logger(), "tra_std_lidar_x " << params_.tra_std_lidar_x);
    RCLCPP_INFO_STREAM(get_logger(), "tra_std_lidar_y " << params_.tra_std_lidar_y);
    RCLCPP_INFO_STREAM(get_logger(), "tra_std_acc " << params_.tra_std_acc);
    RCLCPP_INFO_STREAM(get_logger(), "tra_std_yaw_rate " << params_.tra_std_yaw_rate);
    RCLCPP_INFO_STREAM(get_logger(), "tra_lambda " << params_.tra_lambda);
    RCLCPP_INFO_STREAM(get_logger(), "tra_aging_bad " << params_.tra_aging_bad);
    RCLCPP_INFO_STREAM(get_logger(), "tra_occ_factor " << params_.tra_occ_factor);
    RCLCPP_INFO_STREAM(
        get_logger(), "tra_min_dist_between_tracks " << params_.tra_min_dist_between_tracks);
    RCLCPP_INFO_STREAM(get_logger(), "p_init_x " << params_.p_init_x);
    RCLCPP_INFO_STREAM(get_logger(), "p_init_y " << params_.p_init_y);
    RCLCPP_INFO_STREAM(get_logger(), "p_init_v " << params_.p_init_v);
    RCLCPP_INFO_STREAM(get_logger(), "p_init_yaw " << params_.p_init_yaw);
    RCLCPP_INFO_STREAM(get_logger(), "p_init_yaw_rate " << params_.p_init_yaw_rate);
    RCLCPP_INFO_STREAM(get_logger(), "x_bound " << clustering_params_.x_bound);
    RCLCPP_INFO_STREAM(get_logger(), "y_bound " << clustering_params_.y_bound);
    RCLCPP_INFO_STREAM(get_logger(), "z_bound " << clustering_params_.z_bound);
    RCLCPP_INFO_STREAM(get_logger(), "downsample_voxel_size " << clustering_params_.downsample_voxel_size);
    RCLCPP_INFO_STREAM(get_logger(), "remove_ground_plane_thres " << clustering_params_.remove_ground_plane_thres);
    RCLCPP_INFO_STREAM(get_logger(), "clustering_min_points " << clustering_params_.clustering_min_points);
    RCLCPP_INFO_STREAM(get_logger(), "clustering_max_points " << clustering_params_.clustering_max_points);
    RCLCPP_INFO_STREAM(get_logger(), "clustering_max_step_size " << clustering_params_.clustering_max_step_size);
    RCLCPP_INFO_STREAM(get_logger(), "sacle_up_objects " << clustering_params_.sacle_up_objects);

    buffer_ = std::make_shared<tf2_ros::Buffer>(get_clock());
    transform_listener_ = std::make_shared<tf2_ros::TransformListener>(*buffer_);
    is_initialized_ = false;

    // Measurement covariance
    R_laser_ = Eigen::MatrixXd(params_.tra_dim_z, params_.tra_dim_z);
    R_laser_ << params_.tra_std_lidar_x * params_.tra_std_lidar_x, 0,
        0, params_.tra_std_lidar_y * params_.tra_std_lidar_y;

    // Define weights for UKF
    weights_ = Eigen::VectorXd(2 * params_.tra_dim_x_aug + 1);
    weights_(0) = params_.tra_lambda /
                  (params_.tra_lambda + params_.tra_dim_x_aug);
    for (int i = 1; i < 2 * params_.tra_dim_x_aug + 1; i++)
    {
        weights_(i) = 0.5 / (params_.tra_dim_x_aug + params_.tra_lambda);
    }

    // Start ids for track with 0
    track_id_counter_ = 0;

    // Define Publisher
    list_tracked_objects_pub_ =
        this->create_publisher<vox_nav_msgs::msg::ObjectArray>(
            "/tracking/objects",
            rclcpp::SystemDefaultsQoS());

    // Init counter for publishing
    time_frame_ = 0;
    RCLCPP_INFO(get_logger(), "Creating...");
}

RawCloudClusteringTracking::~RawCloudClusteringTracking()
{
    RCLCPP_INFO(get_logger(), "Destroying...");
}

void RawCloudClusteringTracking::cloudOdomCallback(
    const sensor_msgs::msg::PointCloud2::ConstSharedPtr &cloud,
    const geometry_msgs::msg::PoseArray::ConstSharedPtr &poses)
{
    pcl::PointCloud<pcl::PointXYZRGB>::Ptr pcl_curr(new pcl::PointCloud<pcl::PointXYZRGB>());
    pcl::fromROSMsg(*cloud, *pcl_curr);

    pcl_curr = vox_nav_utilities::crop_box<pcl::PointXYZRGB>(
        pcl_curr,
        Eigen::Vector4f(-clustering_params_.x_bound, -clustering_params_.y_bound, -clustering_params_.z_bound, 1),
        Eigen::Vector4f(clustering_params_.x_bound, clustering_params_.y_bound, clustering_params_.z_bound, 1));
    pcl_curr = vox_nav_utilities::downsampleInputCloud<pcl::PointXYZRGB>(pcl_curr, clustering_params_.downsample_voxel_size);
    pcl_curr = vox_nav_utilities::segmentSurfacePlane<pcl::PointXYZRGB>(pcl_curr, clustering_params_.remove_ground_plane_thres, true);

    pcl_ros::transformPointCloud("map", *pcl_curr, *pcl_curr, *buffer_);

    auto clusters = vox_nav_utilities::euclidean_clustering<pcl::PointXYZRGB>(
        pcl_curr,
        clustering_params_.clustering_min_points,
        clustering_params_.clustering_max_points,
        clustering_params_.clustering_max_step_size);
    vox_nav_utilities::publishClustersCloud(cloud_clusters_pub_, cloud->header, clusters);

    std::vector<std::pair<Eigen::Vector3f, Eigen::Vector3f>> cluster_boxes_vector;
    for (auto &&cluster : clusters)
    {
        auto cupoch_cloud = std::make_shared<cupoch::geometry::PointCloud>();
        thrust::host_vector<Eigen::Vector3f> points;
        for (auto &&i : cluster->points)
        {
            points.push_back(Eigen::Vector3f(i.x, i.y, i.z));
        }
        cupoch_cloud->SetPoints(points);
        auto oobb = cupoch_cloud->GetAxisAlignedBoundingBox();
        cluster_boxes_vector.push_back(std::make_pair(oobb.GetMinBound(), oobb.GetMaxBound()));
    }

    vox_nav_msgs::msg::ObjectArray object_array;

    for (size_t i = 0; i < cluster_boxes_vector.size(); i++)
    {
        vox_nav_msgs::msg::Object object;

        auto mvbb_corners_geometry_msgs = cluster_boxes_vector[i];
        geometry_msgs::msg::PoseStamped object_pose;
        object_pose.header.stamp = cloud->header.stamp;
        object_pose.header.frame_id = "map";
        object_pose.pose.position.x =
            (mvbb_corners_geometry_msgs.second.x() + mvbb_corners_geometry_msgs.first.x()) / 2.0;
        object_pose.pose.position.y =
            (mvbb_corners_geometry_msgs.second.y() + mvbb_corners_geometry_msgs.first.y()) / 2.0;
        object_pose.pose.position.z =
            (mvbb_corners_geometry_msgs.second.z() + mvbb_corners_geometry_msgs.first.z()) / 2.0;

        object.orientation = 0.0;
        object.world_pose.header = object_pose.header;
        object.world_pose.point = object_pose.pose.position;
        object.velo_pose.header = cloud->header;

        geometry_msgs::msg::PoseStamped object_pose_lidar;
        rclcpp::Duration transform_tolerance(0, 500);

        auto result = vox_nav_utilities::transformPose(buffer_,
                                                       cloud->header.frame_id,
                                                       object_pose,
                                                       object_pose_lidar,
                                                       transform_tolerance);
        object.velo_pose.point = object_pose_lidar.pose.position;

        object.height = clustering_params_.sacle_up_objects *
                        std::abs(mvbb_corners_geometry_msgs.second.z() - mvbb_corners_geometry_msgs.first.z());
        object.width = clustering_params_.sacle_up_objects *
                       std::abs(mvbb_corners_geometry_msgs.second.y() - mvbb_corners_geometry_msgs.first.y());
        object.length = clustering_params_.sacle_up_objects *
                        std::abs(mvbb_corners_geometry_msgs.second.x() - mvbb_corners_geometry_msgs.first.x());
        object.id = i;
        object.semantic_id = 0; // assume that we dont know
        object.is_new_track = true;
        object_array.header = object_pose.header;
        object_array.objects.push_back(object);
    }

    auto time_stamp = get_clock()->now();
    if (is_initialized_)
    {
        double dt = (now() - last_time_stamp_).seconds();
        Prediction(dt);
        GlobalNearestNeighbor(object_array);
        Update(object_array);
        TrackManagement(object_array);
    }
    else
    {
        // Initialize tracks
        for (int i = 0; i < object_array.objects.size(); ++i)
        {
            initTrack(object_array.objects[i]);
        }
        is_initialized_ = true;
    }
    last_time_stamp_ = time_stamp;
    time_frame_++;

    publishTracks(cloud->header);
}

void RawCloudClusteringTracking::initTrack(const vox_nav_msgs::msg::Object &obj)
{

    // Only if object can be a track
    if (!obj.is_new_track)
    {
        return;
    }

    // Create new track
    Track track = Track();

    // Add id and increment
    track.id = track_id_counter_;
    track_id_counter_++;

    // Add state information
    track.sta.x = Eigen::VectorXd::Zero(params_.tra_dim_x);
    track.sta.x[0] = obj.world_pose.point.x;
    track.sta.x[1] = obj.world_pose.point.y;
    track.sta.z = obj.world_pose.point.z;
    track.sta.P = Eigen::MatrixXd::Zero(params_.tra_dim_x, params_.tra_dim_x);
    track.sta.P << params_.p_init_x, 0, 0, 0, 0,
        0, params_.p_init_y, 0, 0, 0,
        0, 0, params_.p_init_v, 0, 0,
        0, 0, 0, params_.p_init_yaw, 0,
        0, 0, 0, 0, params_.p_init_yaw_rate;
    track.sta.Xsig_pred = Eigen::MatrixXd::Zero(
        params_.tra_dim_x,
        2 * params_.tra_dim_x_aug + 1);

    // Add semantic information
    track.sem.name = obj.semantic_name;
    track.sem.id = obj.semantic_id;
    track.sem.confidence = obj.semantic_confidence;

    // Add geometric information
    track.geo.width = obj.width;
    track.geo.length = obj.length;
    track.geo.height = obj.height;
    track.geo.orientation = obj.orientation;

    // Add unique color
    std::random_device rd;
    std::mt19937 mt(rd());
    std::uniform_real_distribution<double> dist(0.0, 255.0);
    track.r = dist(mt);
    track.g = dist(mt);
    track.b = dist(mt);
    track.prob_existence = 1.0f;

    track.hist.historic_positions.push_back(
        Eigen::Vector3f(obj.world_pose.point.x,
                        obj.world_pose.point.y,
                        obj.world_pose.point.z));

    // Push back to track list
    tracks_.push_back(track);
}

void RawCloudClusteringTracking::Prediction(const double delta_t)
{

    // Buffer variables
    Eigen::VectorXd x_aug = Eigen::VectorXd(params_.tra_dim_x_aug);
    Eigen::MatrixXd P_aug = Eigen::MatrixXd(params_.tra_dim_x_aug, params_.tra_dim_x_aug);
    Eigen::MatrixXd Xsig_aug = Eigen::MatrixXd(params_.tra_dim_x_aug, 2 * params_.tra_dim_x_aug + 1);

    // Loop through all tracks
    for (int i = 0; i < tracks_.size(); ++i)
    {

        // Grab track
        Track &track = tracks_[i];

        /******************************************************************************
         * 1. Generate augmented sigma points
         */

        // Fill augmented mean state
        x_aug.head(5) = track.sta.x;
        x_aug(5) = 0;
        x_aug(6) = 0;

        // Fill augmented covariance matrix
        P_aug.fill(0.0);
        P_aug.topLeftCorner(5, 5) = track.sta.P;
        P_aug(5, 5) = params_.tra_std_acc * params_.tra_std_acc;
        P_aug(6, 6) = params_.tra_std_yaw_rate * params_.tra_std_yaw_rate;

        // Create square root matrix
        Eigen::MatrixXd L = P_aug.llt().matrixL();

        // Create augmented sigma points
        Xsig_aug.col(0) = x_aug;
        for (int j = 0; j < params_.tra_dim_x_aug; j++)
        {
            Xsig_aug.col(j + 1) = x_aug +
                                  std::sqrt(params_.tra_lambda + params_.tra_dim_x_aug) * L.col(j);
            Xsig_aug.col(j + 1 + params_.tra_dim_x_aug) = x_aug -
                                                          std::sqrt(params_.tra_lambda + params_.tra_dim_x_aug) *
                                                              L.col(j);
        }

        /******************************************************************************
         * 2. Predict sigma points
         */

        for (int j = 0; j < 2 * params_.tra_dim_x_aug + 1; j++)
        {

            // Grab values for better readability
            double p_x = Xsig_aug(0, j);
            double p_y = Xsig_aug(1, j);
            double v = Xsig_aug(2, j);
            double yaw = Xsig_aug(3, j);
            double yawd = Xsig_aug(4, j);
            double nu_a = Xsig_aug(5, j);
            double nu_yawdd = Xsig_aug(6, j);

            // Predicted state values
            double px_p, py_p;

            // Avoid division by zero
            if (fabs(yawd) > 0.001)
            {
                px_p = p_x + v / yawd * (sin(yaw + yawd * delta_t) - sin(yaw));
                py_p = p_y + v / yawd * (cos(yaw) - cos(yaw + yawd * delta_t));
            }
            else
            {
                px_p = p_x + v * delta_t * cos(yaw);
                py_p = p_y + v * delta_t * sin(yaw);
            }
            double v_p = v;
            double yaw_p = yaw + yawd * delta_t;
            double yawd_p = yawd;

            // Add noise
            px_p = px_p + 0.5 * nu_a * delta_t * delta_t * cos(yaw);
            py_p = py_p + 0.5 * nu_a * delta_t * delta_t * sin(yaw);
            v_p = v_p + nu_a * delta_t;
            yaw_p = yaw_p + 0.5 * nu_yawdd * delta_t * delta_t;
            yawd_p = yawd_p + nu_yawdd * delta_t;

            // Write predicted sigma point into right column
            track.sta.Xsig_pred(0, j) = px_p;
            track.sta.Xsig_pred(1, j) = py_p;
            track.sta.Xsig_pred(2, j) = v_p;
            track.sta.Xsig_pred(3, j) = yaw_p;
            track.sta.Xsig_pred(4, j) = yawd_p;
        }

        /******************************************************************************
         * 3. Predict state vector and state covariance
         */
        // Predicted state mean
        track.sta.x.fill(0.0);
        for (int j = 0; j < 2 * params_.tra_dim_x_aug + 1; j++)
        {
            track.sta.x = track.sta.x + weights_(j) *
                                            track.sta.Xsig_pred.col(j);
        }

        // Predicted state covariance matrix
        track.sta.P.fill(0.0);

        // Iterate over sigma points
        for (int j = 0; j < 2 * params_.tra_dim_x_aug + 1; j++)
        {

            // State difference
            Eigen::VectorXd x_diff = track.sta.Xsig_pred.col(j) - track.sta.x;

            // Angle normalization
            while (x_diff(3) > M_PI)
            {
                x_diff(3) -= 2. * M_PI;
            }
            while (x_diff(3) < -M_PI)
            {
                x_diff(3) += 2. * M_PI;
            }

            track.sta.P = track.sta.P + weights_(j) * x_diff *
                                            x_diff.transpose();
        }
    }
}

void RawCloudClusteringTracking::GlobalNearestNeighbor(
    const vox_nav_msgs::msg::ObjectArray &detected_objects)
{

    // Define assoication vectors
    da_tracks_ = std::vector<int>(tracks_.size(), -1);
    da_objects_ = std::vector<int>(detected_objects.objects.size(), -1);

    // Loop through tracks
    for (int i = 0; i < tracks_.size(); ++i)
    {

        // Buffer variables
        std::vector<float> distances;
        std::vector<int> matches;

        // Set data association parameters depending on if
        // the track is a car or a pedestrian
        float gate;
        float box_gate;

        // Pedestrian
        /*if (tracks_[i].sem.id == 11) {
            gate = params_.da_ped_dist_pos;
            box_gate = params_.da_ped_dist_form;
        }
            // Car
        else if (tracks_[i].sem.id == 13) {
            gate = params_.da_car_dist_pos;
            box_gate = params_.da_car_dist_form;
        } else {
            RCLCPP_WARN(get_logger(), "Wrong semantic for track [%d]", tracks_[i].id);
        }*/
        // For now treat every obstacle with pedestrian dynamics
        gate = params_.da_ped_dist_pos;
        box_gate = params_.da_ped_dist_form;

        // Loop through detected objects
        for (int j = 0; j < detected_objects.objects.size(); ++j)
        {

            // Calculate distance between track and detected object
            if (tracks_[i].sem.id == detected_objects.objects[j].semantic_id)
            {
                float dist = CalculateDistance(
                    tracks_[i],
                    detected_objects.objects[j]);

                if (dist < gate)
                {
                    distances.push_back(dist);
                    matches.push_back(j);
                }
            }
        }

        // If track exactly finds one match assign it
        if (matches.size() == 1)
        {

            float box_dist = CalculateEuclideanAndBoxOffset(
                tracks_[i],
                detected_objects.objects[matches[0]]);
            if (box_dist < box_gate)
            {
                da_tracks_[i] = matches[0];
                da_objects_[matches[0]] = i;
            }
        }
        // If found more then take best match and block other measurements
        else if (matches.size() > 1)
        {

            // Block other measurements to NOT be initialized
            RCLCPP_WARN(get_logger(), "Multiple associations for track [%d]", tracks_[i].id);

            // Calculate all box distances and find minimum
            float min_box_dist = box_gate;
            int min_box_index = -1;

            for (int k = 0; k < matches.size(); ++k)
            {

                float box_dist = CalculateEuclideanAndBoxOffset(
                    tracks_[i],
                    detected_objects.objects[matches[k]]);

                if (box_dist < min_box_dist)
                {
                    min_box_index = k;
                    min_box_dist = box_dist;
                }
            }

            for (int k = 0; k < matches.size(); ++k)
            {
                if (k == min_box_index)
                {
                    da_objects_[matches[k]] = i;
                    da_tracks_[i] = matches[k];
                }
                else
                {
                    da_objects_[matches[k]] = -2;
                }
            }
        }
        else
        {
            RCLCPP_WARN(get_logger(), "No measurement found for track [%d]", tracks_[i].id);
        }
    }
}

float RawCloudClusteringTracking::CalculateDistance(
    const Track &track,
    const vox_nav_msgs::msg::Object &object)
{

    // Calculate euclidean distance in x,y,z coordinates of track and object
    return std::abs(track.sta.x(0) - object.world_pose.point.x) +
           std::abs(track.sta.x(1) - object.world_pose.point.y) +
           std::abs(track.sta.z - object.world_pose.point.z);
}

float RawCloudClusteringTracking::CalculateEuclideanDistanceBetweenTracks(
    const Track &t1,
    const Track &t2)
{

    // Calculate euclidean distance in x,y,z coordinates of two tracks
    return sqrt(
        std::pow(t1.sta.x(0) - t2.sta.x(0), 2) +
        std::pow(t1.sta.x(1) - t2.sta.x(1), 2) +
        std::pow(t1.sta.z - t2.sta.z, 2));
}

float RawCloudClusteringTracking::CalculateBoxMismatch(
    const Track &track,
    const vox_nav_msgs::msg::Object &object)
{

    // Calculate mismatch of both tracked cube and detected cube
    float box_wl_switched = std::abs(track.geo.width - object.length) +
                            std::abs(track.geo.length - object.width);
    float box_wl_ordered = std::abs(track.geo.width - object.width) +
                           std::abs(track.geo.length - object.length);
    float box_mismatch = (box_wl_switched < box_wl_ordered) ? box_wl_switched : box_wl_ordered;
    box_mismatch += std::abs(track.geo.height - object.height);
    return box_mismatch;
}

float RawCloudClusteringTracking::CalculateEuclideanAndBoxOffset(
    const Track &track,
    const vox_nav_msgs::msg::Object &object)
{

    // Sum of euclidean offset and box mismatch
    return CalculateDistance(track, object) +
           CalculateBoxMismatch(track, object);
}

bool RawCloudClusteringTracking::compareGoodAge(Track t1, Track t2)
{
    return t1.hist.good_age < t2.hist.good_age;
}

void RawCloudClusteringTracking::Update(const vox_nav_msgs::msg::ObjectArray &detected_objects)
{

    // Buffer variables
    Eigen::VectorXd z = Eigen::VectorXd(params_.tra_dim_z);
    Eigen::MatrixXd Zsig;
    Eigen::VectorXd z_pred = Eigen::VectorXd(params_.tra_dim_z);
    Eigen::MatrixXd S = Eigen::MatrixXd(params_.tra_dim_z, params_.tra_dim_z);
    Eigen::MatrixXd Tc = Eigen::MatrixXd(params_.tra_dim_x, params_.tra_dim_z);

    // Loop through all tracks
    for (int i = 0; i < tracks_.size(); ++i)
    {

        // Grab track
        Track &track = tracks_[i];

        // If track has not found any measurement
        if (da_tracks_[i] == -1)
        {

            // Increment bad aging
            track.hist.bad_age++;
        }
        // If track has found a measurement update it
        else
        {

            // Grab measurement
            z << detected_objects.objects[da_tracks_[i]].world_pose.point.x,
                detected_objects.objects[da_tracks_[i]].world_pose.point.y;

            /******************************************************************************
             * 1. Predict measurement
             */
            // Init measurement sigma points
            Zsig = track.sta.Xsig_pred.topLeftCorner(
                params_.tra_dim_z,
                2 * params_.tra_dim_x_aug + 1);

            // Mean predicted measurement
            z_pred.fill(0.0);
            for (int j = 0; j < 2 * params_.tra_dim_x_aug + 1; j++)
            {
                z_pred = z_pred + weights_(j) * Zsig.col(j);
            }

            S.fill(0.0);
            Tc.fill(0.0);
            for (int j = 0; j < 2 * params_.tra_dim_x_aug + 1; j++)
            {

                // Residual
                Eigen::VectorXd z_sig_diff = Zsig.col(j) - z_pred;
                S = S + weights_(j) * z_sig_diff * z_sig_diff.transpose();

                // State difference
                Eigen::VectorXd x_diff = track.sta.Xsig_pred.col(j) - track.sta.x;

                // Angle normalization
                while (x_diff(3) > M_PI)
                {
                    x_diff(3) -= 2. * M_PI;
                }
                while (x_diff(3) < -M_PI)
                {
                    x_diff(3) += 2. * M_PI;
                }

                Tc = Tc + weights_(j) * x_diff * z_sig_diff.transpose();
            }

            // Add measurement noise covariance matrix
            S = S + R_laser_;

            /******************************************************************************
             * 2. Update state vector and covariance matrix
             */
            // Kalman gain K;
            Eigen::MatrixXd K = Tc * S.inverse();

            // Residual
            Eigen::VectorXd z_diff = z - z_pred;

            // Update state mean and covariance matrix
            track.sta.x = track.sta.x + K * z_diff;
            track.sta.P = track.sta.P - K * S * K.transpose();

            // Update History
            track.hist.good_age++;
            track.hist.bad_age = 0;

            /******************************************************************************
             * 3. Update geometric information of track
             */
            // Calculate area of detection and track
            float det_area =
                detected_objects.objects[da_tracks_[i]].length *
                detected_objects.objects[da_tracks_[i]].width;
            float tra_area = track.geo.length * track.geo.width;

            // If track became strongly smaller keep the shape
            if (params_.tra_occ_factor * det_area < tra_area)
            {
                RCLCPP_WARN(
                    get_logger(), "Track [%d] probably occluded because of dropping size"
                                  " from [%f] to [%f]",
                    track.id, tra_area, det_area);
            }
            // Update the form of the track with measurement
            track.geo.length =
                detected_objects.objects[da_tracks_[i]].length;
            track.geo.width =
                detected_objects.objects[da_tracks_[i]].width;
            track.geo.height =
                detected_objects.objects[da_tracks_[i]].height;

            // Update orientation and ground level
            track.geo.orientation =
                detected_objects.objects[da_tracks_[i]].orientation;
            track.sta.z =
                detected_objects.objects[da_tracks_[i]].world_pose.point.z;

            track.hist.historic_positions.push_back(
                Eigen::Vector3f(detected_objects.objects[da_tracks_[i]].world_pose.point.x,
                                detected_objects.objects[da_tracks_[i]].world_pose.point.y,
                                detected_objects.objects[da_tracks_[i]].world_pose.point.z));
        }
    }
}

void RawCloudClusteringTracking::TrackManagement(const vox_nav_msgs::msg::ObjectArray &detected_objects)
{

    // Delete spuriors tracks
    for (int i = 0; i < tracks_.size(); ++i)
    {

        // Deletion condition
        if (tracks_[i].hist.bad_age >= params_.tra_aging_bad)
        {

            // Print
            RCLCPP_INFO(get_logger(), "Deletion of T [%d]", tracks_[i].id);

            // Swap track with end of vector and pop back
            std::swap(tracks_[i], tracks_.back());
            tracks_.pop_back();
        }
    }

    // Create new ones out of untracked new detected object hypothesis
    // Initialize tracks
    for (int i = 0; i < detected_objects.objects.size(); ++i)
    {

        // Unassigned object condition
        if (da_objects_[i] == -1)
        {

            // Init new track
            initTrack(detected_objects.objects[i]);
        }
    }

    // Sort tracks upon age
    std::sort(
        tracks_.begin(), tracks_.end(), [](Track &t1, Track &t2)
        { return t1.hist.good_age > t2.hist.good_age; });

    // Clear duplicated tracks
    for (int i = tracks_.size() - 1; i >= 0; --i)
    {
        for (int j = i - 1; j >= 0; --j)
        {
            float dist = CalculateEuclideanDistanceBetweenTracks(tracks_[i], tracks_[j]);
            // ROS_INFO("DIST T [%d] and T [%d] = %f ", tracks_[i].id, tracks_[j].id, dist);
            if (dist < params_.tra_min_dist_between_tracks)
            {
                RCLCPP_WARN(
                    get_logger(),
                    "TOO CLOSE: T [%d] and T [%d] = %f ->  T [%d] deleted ",
                    tracks_[i].id, tracks_[j].id, dist, tracks_[i].id);
                std::swap(tracks_[i], tracks_.back());
                tracks_.pop_back();
            }
        }
    }
}

void RawCloudClusteringTracking::publishTracks(const std_msgs::msg::Header &header)
{
    // Create track message
    vox_nav_msgs::msg::ObjectArray track_list;
    track_list.header.stamp = header.stamp;
    track_list.header.frame_id = "map";

    visualization_msgs::msg::MarkerArray marker_array;

    // Loop over all tracks
    for (int i = 0; i < tracks_.size(); ++i)
    {

        // Grab track
        Track &track = tracks_[i];

        // Create new message and fill it
        vox_nav_msgs::msg::Object track_msg;
        track_msg.id = track.id;
        track_msg.world_pose.header.frame_id = "map";
        track_msg.world_pose.point.x = track.sta.x[0];
        track_msg.world_pose.point.y = track.sta.x[1];
        track_msg.world_pose.point.z = track.sta.z;

        try
        {
            buffer_->transform(
                track_msg.world_pose,
                track_msg.cam_pose,
                "base_link");
            buffer_->transform(
                track_msg.world_pose,
                track_msg.velo_pose,
                "base_link");
        }
        catch (tf2::TransformException &ex)
        {
            RCLCPP_ERROR(get_logger(), "Received an exception trying to transform a point from"
                                       "\"base_link\" to \"map\": %s",
                         ex.what());
        }
        track_msg.heading = track.sta.x[3];
        track_msg.velocity = track.sta.x[2];
        track_msg.width = track.geo.width;
        track_msg.length = track.geo.length;
        track_msg.height = track.geo.height;
        track_msg.orientation = track.geo.orientation;
        track_msg.semantic_name = track.sem.name;
        track_msg.semantic_id = track.sem.id;
        track_msg.semantic_confidence = track.sem.confidence;
        track_msg.r = track.r;
        track_msg.g = track.g;
        track_msg.b = track.b;
        track_msg.a = track.prob_existence;

        // Push back track message
        track_list.objects.push_back(track_msg);

        VizObject viz_obj;
        // Fill in bounding box information
        viz_obj.bb.action = visualization_msgs::msg::Marker::ADD;
        viz_obj.bb.ns = "my_namespace";
        viz_obj.bb.type = visualization_msgs::msg::Marker::CYLINDER;
        viz_obj.bb.header.frame_id = "map";
        viz_obj.bb.lifetime = rclcpp::Duration(1.0);
        viz_obj.bb.id = i;
        viz_obj.bb.pose.position.x = track_msg.world_pose.point.x;
        viz_obj.bb.pose.position.y = track_msg.world_pose.point.y;
        viz_obj.bb.pose.position.z = track_msg.world_pose.point.z;
        viz_obj.bb.pose.orientation = vox_nav_utilities::getMsgQuaternionfromRPY(0, 0, track_msg.orientation);
        viz_obj.bb.scale.x = track_msg.length;
        viz_obj.bb.scale.y = track_msg.width;
        viz_obj.bb.scale.z = track_msg.height;
        viz_obj.bb.color.a = 0.75;
        viz_obj.bb.color.r = float(track_msg.r) / 255.0;
        viz_obj.bb.color.g = float(track_msg.g) / 255.0;
        viz_obj.bb.color.b = float(track_msg.b) / 255.0;

        double direction_angle = 0.0;
        // Fill in arrow information
        if (track.hist.historic_positions.size() > 3)
        {
            int num_elements = track.hist.historic_positions.size();

            double dy = track.hist.historic_positions.back().y() -
                        track.hist.historic_positions[num_elements - 3].y();

            double dx = track.hist.historic_positions.back().x() -
                        track.hist.historic_positions[num_elements - 3].x();

            direction_angle = std::atan2(dy, dx);

            viz_obj.arr.action = visualization_msgs::msg::Marker::ADD;
            viz_obj.arr.ns = "my_namespace";
            viz_obj.arr.type = visualization_msgs::msg::Marker::ARROW;
            viz_obj.arr.header.frame_id = "map";
            viz_obj.arr.lifetime = rclcpp::Duration(1.0);
            viz_obj.arr.id = i + 100;
            viz_obj.arr.scale.x = 0.2;
            viz_obj.arr.scale.y = 0.35;
            viz_obj.arr.scale.z = 0.15;
            geometry_msgs::msg::Point arr_start, arr_end;
            arr_start.x = track.hist.historic_positions.back().x();
            arr_start.y = track.hist.historic_positions.back().y();
            arr_start.z = track.hist.historic_positions.back().z();
            arr_end.x = track.hist.historic_positions.back().x() + dx;
            arr_end.y = track.hist.historic_positions.back().y() + dy;
            arr_end.z = track.hist.historic_positions.back().z();
            viz_obj.arr.points.push_back(arr_start);
            viz_obj.arr.points.push_back(arr_end);
            std_msgs::msg::ColorRGBA color;
            color.a = 0.75;
            color.r = float(track_msg.r) / 255.0;
            color.g = float(track_msg.g) / 255.0;
            color.b = float(track_msg.b) / 255.0;
            viz_obj.arr.colors.push_back(color);
            viz_obj.arr.colors.push_back(color);

            viz_obj.arr.color.a = 0.75;
            viz_obj.arr.color.r = float(track_msg.r) / 255.0;
            viz_obj.arr.color.g = float(track_msg.g) / 255.0;
            viz_obj.arr.color.b = float(track_msg.b) / 255.0;
        }

        visualization_msgs::msg::Marker dynamic_obj_waypoints;
        dynamic_obj_waypoints.action = visualization_msgs::msg::Marker::ADD;
        dynamic_obj_waypoints.ns = "my_namespace";
        dynamic_obj_waypoints.type = visualization_msgs::msg::Marker::SPHERE_LIST;
        dynamic_obj_waypoints.header.frame_id = "map";
        dynamic_obj_waypoints.lifetime = rclcpp::Duration(1.0);
        dynamic_obj_waypoints.id = i + 500;
        dynamic_obj_waypoints.scale.x = 0.2;
        dynamic_obj_waypoints.scale.y = 0.2;
        dynamic_obj_waypoints.scale.z = 0.2;
        if (abs(track_msg.velocity) > 0.1)
        {
            std_msgs::msg::ColorRGBA color;
            color.r = 1.0;
            color.a = 0.8;
            for (auto t : track.hist.historic_positions)
            {
                geometry_msgs::msg::Point point;
                point.x = t.x();
                point.y = t.y();
                point.z = t.z();
                dynamic_obj_waypoints.points.push_back(point);
                dynamic_obj_waypoints.colors.push_back(color);
            }
        }

        // Fill in text information
        viz_obj.txt.action = visualization_msgs::msg::Marker::ADD;
        viz_obj.txt.ns = "my_namespace";
        viz_obj.txt.type = visualization_msgs::msg::Marker::TEXT_VIEW_FACING;
        viz_obj.txt.header.frame_id = "map";
        viz_obj.txt.lifetime = rclcpp::Duration(1.0);
        viz_obj.txt.id = i + 200;
        viz_obj.txt.pose.position.x = track_msg.world_pose.point.x;
        viz_obj.txt.pose.position.y = track_msg.world_pose.point.y;
        viz_obj.txt.pose.position.z = track_msg.world_pose.point.z + track_msg.height;
        viz_obj.txt.scale.x = 1.0;
        viz_obj.txt.scale.y = 1.0;
        viz_obj.txt.scale.z = 1.0;
        viz_obj.txt.color.a = 1.0;
        viz_obj.txt.color.r = float(track_msg.r) / 255.0;
        viz_obj.txt.color.g = float(track_msg.g) / 255.0;
        viz_obj.txt.color.b = float(track_msg.b) / 255.0;
        viz_obj.txt.text = std::to_string(track_msg.id);

        marker_array.markers.push_back(viz_obj.arr);
        marker_array.markers.push_back(viz_obj.bb);
        marker_array.markers.push_back(viz_obj.txt);
        marker_array.markers.push_back(dynamic_obj_waypoints);
    }

    // Print
    RCLCPP_INFO(get_logger(), "Publishing Tracking [%d]: # Tracks [%d]", time_frame_,
                int(tracks_.size()));

    // Publish
    list_tracked_objects_pub_->publish(track_list);
    tracking_markers_pub_->publish(marker_array);
}

int main(int argc, char const *argv[])
{
    rclcpp::init(argc, argv);
    auto node = std::make_shared<RawCloudClusteringTracking>();
    rclcpp::spin(node);
    rclcpp::shutdown();
    return 0;
}
