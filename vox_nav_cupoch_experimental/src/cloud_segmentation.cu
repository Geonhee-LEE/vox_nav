#include "hip/hip_runtime.h"
// Copyright (c) 2020 Fetullah Atas, Norwegian University of Life Sciences
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "vox_nav_cupoch_experimental/cloud_segmentation.hpp"

struct printf_functor {
    printf_functor(
            const double *radius,
            const int *max_nn,
            const thrust::host_vector<Eigen::Vector3f> *a_colors,
            const cupoch::geometry::KDTreeFlann *kdtree,
            int *counter,
            thrust::host_vector<Eigen::Vector3f> *points,
            thrust::host_vector<Eigen::Vector3f> *colors
    )
            : radius_(radius),
              max_nn_(max_nn),
              a_colors_(a_colors),
              kdtree_(kdtree),
              counter_(counter),
              points_(points),
              colors_(colors) {}

    const double *radius_;
    const int *max_nn_;
    const thrust::host_vector<Eigen::Vector3f> *a_colors_;
    const cupoch::geometry::KDTreeFlann *kdtree_;
    int *counter_;
    thrust::host_vector<Eigen::Vector3f> *points_;
    thrust::host_vector<Eigen::Vector3f> *colors_;

    __host__ __device__
    void operator()(Eigen::Vector3f &i) {
        thrust::host_vector<int> indices;
        thrust::host_vector<float> distance2;
        int k = kdtree_->SearchRadius(
                i, *radius_, 100,
                indices, distance2);
        if (k < *max_nn_) {
            points_->push_back(i);
            colors_->push_back((*a_colors_)[*counter_]);
        }
        ++*counter_;
    }
};

CloudSegmentation::CloudSegmentation()
        : Node("dynamic_points_node"), recieved_first_(false) {
    cloud_subscriber_.subscribe(this, "points", rmw_qos_profile_sensor_data);
    odom_subscriber_.subscribe(this, "odom", rmw_qos_profile_sensor_data);
    imu_subscriber_.subscribe(this, "imu", rmw_qos_profile_sensor_data);

    declare_parameter("dt", 0.0);
    get_parameter("dt", dt_);

    declare_parameter("sensor_height", 0.0);
    get_parameter("sensor_height", sensor_height_);

    cloud_odom_data_approx_time_syncher_.reset(
            new CloudOdomApprxTimeSyncer(
                    CloudOdomApprxTimeSyncPolicy(500),
                    cloud_subscriber_,
                    odom_subscriber_));

    cloud_odom_data_approx_time_syncher_->registerCallback(
            std::bind(
                    &CloudSegmentation::cloudOdomCallback, this,
                    std::placeholders::_1,
                    std::placeholders::_2));

    cloud_pub_ = this->create_publisher<sensor_msgs::msg::PointCloud2>(
            "merged", rclcpp::SystemDefaultsQoS());

    marker_pub_ = this->create_publisher<visualization_msgs::msg::MarkerArray>(
            "correspondings", rclcpp::SystemDefaultsQoS());

    last_odom_msg_ = std::make_shared<nav_msgs::msg::Odometry>();
    last_dynamic_pointcloud_cupoch_ = std::make_shared<cupoch::geometry::PointCloud>();

    last_recieved_msg_stamp_ = now();
}

CloudSegmentation::~CloudSegmentation() {}

void CloudSegmentation::cloudOdomCallback(
        const sensor_msgs::msg::PointCloud2::ConstSharedPtr &cloud,
        const nav_msgs::msg::Odometry::ConstSharedPtr &odom) {
    if (!recieved_first_) {
        recieved_first_ = true;
        last_recieved_msg_stamp_ = cloud->header.stamp;
    }

    pcl::PointCloud<pcl::PointXYZI>::Ptr pcl_curr(new pcl::PointCloud<pcl::PointXYZI>());
    pcl::fromROSMsg(*cloud, *pcl_curr);

    pcl_curr = vox_nav_utilities::crop_box<pcl::PointXYZI>(
            pcl_curr,
            Eigen::Vector4f(-15, -15, -2, 1),
            Eigen::Vector4f(15, 15, 2, 1));

    pcl_curr = vox_nav_utilities::downsampleInputCloud<pcl::PointXYZI>(
            pcl_curr, 0.05);

    pcl::PointCloud<pcl::PointXYZRGB>::Ptr ground_points_pcl(new pcl::PointCloud<pcl::PointXYZRGB>());
    pcl::PointCloud<pcl::PointXYZRGB>::Ptr dynamic_points_pcl(new pcl::PointCloud<pcl::PointXYZRGB>());
    pcl::PointCloud<pcl::PointXYZRGB>::Ptr static_points_pcl(new pcl::PointCloud<pcl::PointXYZRGB>());
    auto static_points_cupoch = std::make_shared<cupoch::geometry::PointCloud>();
    auto dynamic_points_cupoch = std::make_shared<cupoch::geometry::PointCloud>();
    thrust::host_vector<Eigen::Vector3f> static_points, static_colors;
    thrust::host_vector<Eigen::Vector3f> dynamic_points, dynamic_colors;

    auto green_color = vox_nav_utilities::getColorByIndexEig(1);
    auto orange_color = vox_nav_utilities::getColorByIndexEig(5);
    auto yellow_color = vox_nav_utilities::getColorByIndexEig(10);

    for (auto &&h: pcl_curr->points) {
        int this_point_label = static_cast<int>( h.intensity * 255.0);  // labels are burried into intensity
        pcl::PointXYZRGB point;
        point.x = h.x;
        point.y = h.y;
        point.z = h.z;
        Eigen::Vector3f point_eig(point.x, point.y, point.z);
        if (this_point_label == 40) { // ground point label
            ground_points_pcl->points.push_back(point);
        } else if (this_point_label == 30 ||
                   this_point_label == 10)        // person/car point label
        {
            dynamic_points_pcl->points.push_back(point);
        } else { // static obstacle point label
            static_points.push_back(point_eig);
            static_colors.push_back(green_color);
            static_points_pcl->points.push_back(point);
        }
    }

    dynamic_points_pcl =
            vox_nav_utilities::remove_points_within_ground_plane_of_other_cloud<pcl::PointXYZRGB>(
                    dynamic_points_pcl, ground_points_pcl, 0.4);

    dynamic_points_pcl = vox_nav_utilities::denoise_segmented_cloud<pcl::PointXYZRGB>(
            dynamic_points_pcl,
            static_points_pcl, 0.2, 2);

    for (int i = 0; i < dynamic_points_pcl->points.size(); ++i) {
        auto p = dynamic_points_pcl->points[i];
        Eigen::Vector3f point_eig(p.x, p.y, p.z);
        dynamic_points.push_back(point_eig);
        dynamic_colors.push_back(yellow_color);
    }

    dynamic_points_cupoch->SetPoints(dynamic_points);
    dynamic_points_cupoch->SetColors(dynamic_colors);
    static_points_cupoch->SetPoints(static_points);
    static_points_cupoch->SetColors(static_colors);

    rclcpp::Time crr_stamp = cloud->header.stamp;

    if ((crr_stamp - last_recieved_msg_stamp_).seconds() > dt_) {

        auto odom_T = getTransfromfromConsecutiveOdoms(
                std::make_shared<nav_msgs::msg::Odometry>(*odom), last_odom_msg_);
        last_dynamic_pointcloud_cupoch_->Transform(odom_T.inverse());
        last_dynamic_pointcloud_cupoch_->PaintUniformColor(orange_color);

        auto static_and_dynamic_obstacle_cloud =
                *dynamic_points_cupoch + *static_points_cupoch + *last_dynamic_pointcloud_cupoch_;
        auto static_and_dynamic_obstacle_cloud_ptr = std::make_shared<cupoch::geometry::PointCloud>(
                static_and_dynamic_obstacle_cloud);
        using std::chrono::duration;
        using std::chrono::duration_cast;
        using std::chrono::high_resolution_clock;
        using std::chrono::milliseconds;

        auto t1 = high_resolution_clock::now();
        determineObjectMovements(dynamic_points_cupoch, last_dynamic_pointcloud_cupoch_, cloud->header);

        auto t2 = high_resolution_clock::now();
        auto ms_int = duration_cast<milliseconds>(t2 - t1);
        RCLCPP_INFO(get_logger(), "determineObjectMovements take ms %d", ms_int.count());

        sensor_msgs::msg::PointCloud2 denoised_cloud_msg;
        cupoch_conversions::cupochToRos(
                static_and_dynamic_obstacle_cloud_ptr,
                denoised_cloud_msg,
                cloud->header.frame_id);
        denoised_cloud_msg.header = cloud->header;
        cloud_pub_->publish(denoised_cloud_msg);
        last_recieved_msg_stamp_ = cloud->header.stamp;
        last_odom_msg_ = std::make_shared<nav_msgs::msg::Odometry>(*odom);
        last_dynamic_pointcloud_cupoch_ =
                std::make_shared<cupoch::geometry::PointCloud>(*dynamic_points_cupoch);


    }
}

void CloudSegmentation::determineObjectMovements(
        std::shared_ptr<cupoch::geometry::PointCloud> a,
        std::shared_ptr<cupoch::geometry::PointCloud> b,
        std_msgs::msg::Header header) {
    if (!a->points_.size() || !b->points_.size()) {
        RCLCPP_INFO(
                get_logger(),
                "Oneof the cloud is empty Clouds have a: %d b: %d points, object movement cannot be determined",
                a->points_.size(), b->points_.size());
        return;
    }
    RCLCPP_INFO(
            get_logger(),
            "Clouds have a: %d b: %d points",
            a->points_.size(), b->points_.size());

    auto a_points = a->GetPoints();
    auto b_points = b->GetPoints();

    cupoch::utility::device_vector<int> a_clusters = a->ClusterDBSCAN(0.2, 8, false);
    cupoch::utility::device_vector<int> b_clusters = b->ClusterDBSCAN(0.2, 8, false);

    std::map<int, thrust::host_vector<Eigen::Vector3f>> a_cluster_set, b_cluster_set;

    clusterIndices2ClusterSet(a_clusters, a_points, a_cluster_set);
    clusterIndices2ClusterSet(b_clusters, b_points, b_cluster_set);

    std::vector<std::shared_ptr<cupoch::geometry::PointCloud>> a_cluster_vector;
    std::vector<std::shared_ptr<cupoch::geometry::PointCloud>> b_cluster_vector;

    clusterSet2CloudVector(a_cluster_set, a_cluster_vector);
    clusterSet2CloudVector(b_cluster_set, b_cluster_vector);

    visualization_msgs::msg::MarkerArray marker_array;
    visualization_msgs::msg::Marker marker;
    marker.header = header;
    marker.type = visualization_msgs::msg::Marker::SPHERE_LIST;
    marker.action = visualization_msgs::msg::Marker::ADD;
    marker.lifetime = rclcpp::Duration::from_seconds(0);
    marker.scale.x = 0.8;
    marker.scale.y = 0.8;
    marker.scale.z = 0.8;

    for (int i = 0; i < a_cluster_vector.size(); ++i) {
        auto oriented_bbx = a_cluster_vector[i]->GetOrientedBoundingBox();
        geometry_msgs::msg::Point center_point;
        center_point.x = oriented_bbx.GetCenter().x();
        center_point.y = oriented_bbx.GetCenter().y();
        center_point.z = oriented_bbx.GetCenter().z();
        std_msgs::msg::ColorRGBA color;
        color.r = 1.0;
        color.a = 1.0;
        marker.colors.push_back(color);
        marker.points.push_back(center_point);
    }
    for (int i = 0; i < b_cluster_vector.size(); ++i) {
        auto oriented_bbx = b_cluster_vector[i]->GetOrientedBoundingBox();
        geometry_msgs::msg::Point center_point;
        center_point.x = oriented_bbx.GetCenter().x();
        center_point.y = oriented_bbx.GetCenter().y();
        center_point.z = oriented_bbx.GetCenter().z();
        std_msgs::msg::ColorRGBA color;
        color.b = 1.0;
        color.a = 1.0;
        marker.colors.push_back(color);
        marker.points.push_back(center_point);
    }
    marker_array.markers.push_back(marker);
    marker_pub_->publish(marker_array);
}

void CloudSegmentation::clusterSet2CloudVector(
        const std::map<int, thrust::host_vector<Eigen::Vector3f>> &cluster_set,
        std::vector<std::shared_ptr<cupoch::geometry::PointCloud>> &cluster_vector) {
    for (auto it = cluster_set.begin(); it != cluster_set.end(); ++it) {
        if (!it->second.size()) {
            continue;
        }
        RCLCPP_INFO(
                get_logger(),
                "B cluster label %d have : %d points",
                it->first, it->second.size());
        auto this_cluster = std::make_shared<cupoch::geometry::PointCloud>();
        this_cluster->SetPoints(it->second);
        cluster_vector.push_back(this_cluster);
    }
}

void CloudSegmentation::clusterIndices2ClusterSet(
        const cupoch::utility::device_vector<int> &clusters,
        const thrust::host_vector<Eigen::Vector3f> &points,
        std::map<int, thrust::host_vector<Eigen::Vector3f>> &cluster_set) {
    for (int i = 0; i < clusters.size(); ++i) {
        if (clusters[i] < 0) {
            continue;
        }
        auto it = cluster_set.find(clusters[i]);
        if (it != cluster_set.end()) {
            it->second.push_back(points[i]);
        } else {
            cluster_set.insert(
                    std::pair<int, thrust::host_vector<Eigen::Vector3f>>(
                            clusters[i],
                            thrust::host_vector<Eigen::Vector3f>()));
        }
    }
}

std::shared_ptr<cupoch::geometry::PointCloud> CloudSegmentation::denoiseCupochCloud(
        std::shared_ptr<cupoch::geometry::PointCloud> a,
        const std::shared_ptr<cupoch::geometry::PointCloud> b,
        double radius,
        int max_nn) {
    auto denoised_cupoch_cloud = std::make_shared<cupoch::geometry::PointCloud>();
    cupoch::geometry::KDTreeFlann kdtree(*b);
    auto a_points = a->GetPoints();
    auto a_colors = a->GetColors();
    thrust::host_vector<Eigen::Vector3f> points, colors;
    int counter = 0;
    printf_functor f(&radius, &max_nn, &a_colors, &kdtree, &counter, &points, &colors);
    thrust::for_each(thrust::host, a_points.begin(), a_points.end(), f);
    denoised_cupoch_cloud->SetPoints(points);
    denoised_cupoch_cloud->SetColors(colors);
    return denoised_cupoch_cloud;
}

Eigen::Matrix4f CloudSegmentation::getTransfromfromConsecutiveOdoms(
        const nav_msgs::msg::Odometry::SharedPtr a,
        const nav_msgs::msg::Odometry::SharedPtr b) {
    auto traveled_distance =
            Eigen::Vector3f(
                    a->pose.pose.position.x - b->pose.pose.position.x,
                    a->pose.pose.position.y - b->pose.pose.position.y,
                    a->pose.pose.position.z - b->pose.pose.position.z)
                    .norm();

    double yaw_latest, pitch_latest, roll_latest;
    double yaw, pitch, roll;

    vox_nav_utilities::getRPYfromMsgQuaternion(
            a->pose.pose.orientation, roll_latest, pitch_latest, yaw_latest);
    vox_nav_utilities::getRPYfromMsgQuaternion(
            b->pose.pose.orientation, roll, pitch, yaw);

    auto rot = cupoch::geometry::GetRotationMatrixFromXYZ(
            Eigen::Vector3f(
                    roll_latest - roll, pitch_latest - pitch, yaw_latest - yaw));

    auto trans =
            Eigen::Vector3f(
                    traveled_distance * cos(yaw_latest - yaw),
                    traveled_distance * sin(yaw_latest - yaw), sensor_height_);

    Eigen::Matrix4f odom_T = Eigen::Matrix4f::Identity();

    odom_T.block<3, 3>(0, 0) = rot;
    odom_T.block<3, 1>(0, 3) = trans;

    return odom_T;
}

int main(int argc, char const *argv[]) {
    rclcpp::init(argc, argv);
    cupoch::utility::InitializeAllocator();
    auto node = std::make_shared<CloudSegmentation>();
    rclcpp::spin(node);
    rclcpp::shutdown();
    return 0;
}
