#include "hip/hip_runtime.h"
// Copyright (c) 2020 Fetullah Atas, Norwegian University of Life Sciences
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "vox_nav_cupoch_experimental/cloud_segmentation.hpp"

CloudSegmentation::CloudSegmentation()
    : Node("dynamic_points_node"), recieved_first_(false)
{
  cloud_subscriber_.subscribe(this, "points", rmw_qos_profile_sensor_data);
  odom_subscriber_.subscribe(this, "odom", rmw_qos_profile_sensor_data);
  imu_subscriber_.subscribe(this, "imu", rmw_qos_profile_sensor_data);

  declare_parameter("dt", 0.0);
  get_parameter("dt", dt_);

  declare_parameter("sensor_height", 0.0);
  get_parameter("sensor_height", sensor_height_);

  cloud_odom_data_approx_time_syncher_.reset(
      new CloudOdomApprxTimeSyncer(
          CloudOdomApprxTimeSyncPolicy(500),
          cloud_subscriber_,
          odom_subscriber_));

  cloud_odom_data_approx_time_syncher_->registerCallback(
      std::bind(
          &CloudSegmentation::cloudOdomCallback, this,
          std::placeholders::_1,
          std::placeholders::_2));

  cloud_pub_ = this->create_publisher<sensor_msgs::msg::PointCloud2>(
      "merged", rclcpp::SystemDefaultsQoS());

  marker_pub_ = this->create_publisher<visualization_msgs::msg::MarkerArray>(
      "correspondings", rclcpp::SystemDefaultsQoS());

  last_odom_msg_ = std::make_shared<nav_msgs::msg::Odometry>();
  last_dynamic_pointcloud_cupoch_ = std::make_shared<cupoch::geometry::PointCloud>();

  last_recieved_msg_stamp_ = now();
}

CloudSegmentation::~CloudSegmentation() {}

void CloudSegmentation::cloudOdomCallback(
    const sensor_msgs::msg::PointCloud2::ConstSharedPtr &cloud,
    const nav_msgs::msg::Odometry::ConstSharedPtr &odom)
{
  if (!recieved_first_)
  {
    recieved_first_ = true;
    last_recieved_msg_stamp_ = cloud->header.stamp;
  }

  cupoch::utility::device_vector<size_t> ground_points_indices, dynamic_points_indices, static_points_indices;
  pcl::PointCloud<pcl::PointXYZRGB>::Ptr ground_points_pcl(new pcl::PointCloud<pcl::PointXYZRGB>());
  pcl::PointCloud<pcl::PointXYZRGB>::Ptr dynamic_points_pcl(new pcl::PointCloud<pcl::PointXYZRGB>());
  pcl::PointCloud<pcl::PointXYZRGB>::Ptr static_points_pcl(new pcl::PointCloud<pcl::PointXYZRGB>());

  thrust::host_vector<Eigen::Vector3f> points, colors;
  size_t label_counter = 0;
  auto color = vox_nav_utilities::getColorByIndexEig(5);
  sensor_msgs::PointCloud2ConstIterator<float> iter_label(*cloud, "x");
  for (; (iter_label != iter_label.end()); ++iter_label)
  {
    pcl::PointXYZRGB point;
    point.x = iter_label[0];
    point.y = iter_label[1];
    point.z = iter_label[2];
    point.r = color.x();
    point.g = color.y();
    point.b = color.z();

    points.push_back(Eigen::Vector3f(point.x, point.y, point.z));
    colors.push_back(Eigen::Vector3f(point.r, point.g, point.b));

    int this_point_label = iter_label[3];
    if (this_point_label == 40) // ground point label
    {
      ground_points_indices.push_back(label_counter);
      ground_points_pcl->points.push_back(point);
    }
    else if (this_point_label == 30 ||
             this_point_label == 10) // person/car point label
    {
      dynamic_points_indices.push_back(label_counter);
      dynamic_points_pcl->points.push_back(point);
    }
    else // static obstacle point label
    {
      static_points_indices.push_back(label_counter);
      static_points_pcl->points.push_back(point);
    }

    label_counter++;
  }

  cupoch::geometry::PointCloud obstacle_cloud_cupoch;
  obstacle_cloud_cupoch.SetPoints(points);
  obstacle_cloud_cupoch.SetColors(colors);
  auto static_points_cupoch =
      obstacle_cloud_cupoch.SelectByIndex(static_points_indices, false);
  auto ground_points_cupoch =
      obstacle_cloud_cupoch.SelectByIndex(ground_points_indices, false);

  dynamic_points_pcl =
      vox_nav_utilities::remove_points_within_ground_plane_of_other_cloud<pcl::PointXYZRGB>(
          dynamic_points_pcl, ground_points_pcl, 0.4);

  points.clear();
  colors.clear();
  for (int i = 0; i < dynamic_points_pcl->points.size(); ++i)
  {
    auto p = dynamic_points_pcl->points[i];
    points.push_back(Eigen::Vector3f(p.x, p.y, p.z));
    auto c = vox_nav_utilities::getColorByIndexEig(5);
    colors.push_back(Eigen::Vector3f(c.x(), c.y(), c.z()));
  }

  auto dynamic_points_cupoch = std::make_shared<cupoch::geometry::PointCloud>();
  dynamic_points_cupoch->SetPoints(points);
  dynamic_points_cupoch->SetColors(colors);

  dynamic_points_cupoch->PaintUniformColor(
      vox_nav_utilities::getColorByIndexEig(10));
  static_points_cupoch->PaintUniformColor(
      vox_nav_utilities::getColorByIndexEig(1));

  Eigen::Matrix<float, 3, 1> min(-20, -20, -2);
  Eigen::Matrix<float, 3, 1> max(20, 20, 2);
  cupoch::geometry::AxisAlignedBoundingBox<3> bbx(min, max);

  dynamic_points_cupoch = dynamic_points_cupoch->Crop(bbx);
  static_points_cupoch = static_points_cupoch->Crop(bbx);

  dynamic_points_cupoch = dynamic_points_cupoch->VoxelDownSample(0.05);
  static_points_cupoch = static_points_cupoch->VoxelDownSample(0.05);

  dynamic_points_cupoch = denoiseCupochCloud(dynamic_points_cupoch, static_points_cupoch, 0.5, 4);

  rclcpp::Time crr_stamp = cloud->header.stamp;
  if ((crr_stamp - last_recieved_msg_stamp_).seconds() > dt_)
  {

    auto odom_T = getTransfromfromConsecutiveOdoms(std::make_shared<nav_msgs::msg::Odometry>(*odom), last_odom_msg_);
    last_dynamic_pointcloud_cupoch_->Transform(odom_T.inverse());
    last_dynamic_pointcloud_cupoch_->PaintUniformColor(vox_nav_utilities::getColorByIndexEig(5));

    auto static_and_dynamic_obstacle_cloud =
        *dynamic_points_cupoch + *static_points_cupoch + *last_dynamic_pointcloud_cupoch_;
    auto static_and_dynamic_obstacle_cloud_ptr = std::make_shared<cupoch::geometry::PointCloud>(static_and_dynamic_obstacle_cloud);

    determineObjectMovements(dynamic_points_cupoch, last_dynamic_pointcloud_cupoch_, cloud->header);

    sensor_msgs::msg::PointCloud2 denoised_cloud_msg;
    cupoch_conversions::cupochToRos(
        static_and_dynamic_obstacle_cloud_ptr,
        denoised_cloud_msg,
        cloud->header.frame_id);
    denoised_cloud_msg.header = cloud->header;
    cloud_pub_->publish(denoised_cloud_msg);
    last_recieved_msg_stamp_ = cloud->header.stamp;
    last_odom_msg_ = std::make_shared<nav_msgs::msg::Odometry>(*odom);
    last_dynamic_pointcloud_cupoch_ =
        std::make_shared<cupoch::geometry::PointCloud>(*dynamic_points_cupoch);
  }
}

void CloudSegmentation::determineObjectMovements(
    std::shared_ptr<cupoch::geometry::PointCloud> a,
    std::shared_ptr<cupoch::geometry::PointCloud> b,
    std_msgs::msg::Header header)
{

  if (!a->points_.size() || !b->points_.size())
  {
    RCLCPP_WARN(
        get_logger(),
        "One of the cloud is empty, clouds have a: %d b: %d points",
        a->points_.size(), b->points_.size());
    return;
  }

  // REMOVE THE NOISE
  auto denoised_a = a->RemoveStatisticalOutliers(10, 0.1);
  auto denoised_b = b->RemoveStatisticalOutliers(10, 0.1);
  denoised_a = std::get<0>(denoised_a)->RemoveRadiusOutliers(8, 0.2);
  denoised_b = std::get<0>(denoised_b)->RemoveRadiusOutliers(8, 0.2);
  a = std::get<0>(denoised_a);
  b = std::get<0>(denoised_b);

  auto a_points = a->GetPoints();
  auto b_points = b->GetPoints();

  cupoch::utility::device_vector<int> a_clusters = a->ClusterDBSCAN(0.2, 8, false);
  cupoch::utility::device_vector<int> b_clusters = b->ClusterDBSCAN(0.2, 8, false);

  std::map<int, thrust::host_vector<Eigen::Vector3f>> a_cluster_set, b_cluster_set;

  for (int i = 0; i < a_clusters.size(); ++i)
  {
    if (a_clusters[i] < 0)
    {
      continue;
    }
    auto it = a_cluster_set.find(a_clusters[i]);
    if (it != a_cluster_set.end())
    {
      it->second.push_back(a_points[i]);
    }
    else
    {
      a_cluster_set.insert(
          std::pair<int, thrust::host_vector<Eigen::Vector3f>>(
              a_clusters[i],
              thrust::host_vector<Eigen::Vector3f>()));
    }
  }

  for (int i = 0; i < b_clusters.size(); ++i)
  {
    if (b_clusters[i] < 0)
    {
      continue;
    }
    auto it = b_cluster_set.find(b_clusters[i]);
    if (it != b_cluster_set.end())
    {
      it->second.push_back(b_points[i]);
    }
    else
    {
      b_cluster_set.insert(
          std::pair<int, thrust::host_vector<Eigen::Vector3f>>(
              b_clusters[i],
              thrust::host_vector<Eigen::Vector3f>()));
    }
  }

  std::vector<std::shared_ptr<cupoch::geometry::PointCloud>> a_cluster_vector;
  std::vector<std::shared_ptr<cupoch::geometry::PointCloud>> b_cluster_vector;

  for (auto it = a_cluster_set.begin(); it != a_cluster_set.end(); ++it)
  {
    if (!it->second.size())
    {
      continue;
    }
    RCLCPP_INFO(
        get_logger(),
        "A cluster label %d have : %d points",
        it->first, it->second.size());

    auto this_cluster = std::make_shared<cupoch::geometry::PointCloud>();
    this_cluster->SetPoints(it->second);
    a_cluster_vector.push_back(this_cluster);
  }

  for (auto it = b_cluster_set.begin(); it != b_cluster_set.end(); ++it)
  {
    if (!it->second.size())
    {
      continue;
    }
    RCLCPP_INFO(
        get_logger(),
        "B cluster label %d have : %d points",
        it->first, it->second.size());
    auto this_cluster = std::make_shared<cupoch::geometry::PointCloud>();
    this_cluster->SetPoints(it->second);
    b_cluster_vector.push_back(this_cluster);
  }

  RCLCPP_INFO(
      get_logger(),
      "Clouds have a: %d b: %d points",
      a->points_.size(), b->points_.size());

  visualization_msgs::msg::MarkerArray marker_array;
  visualization_msgs::msg::Marker marker;
  marker.header = header;
  marker.type = visualization_msgs::msg::Marker::SPHERE_LIST;
  marker.action = visualization_msgs::msg::Marker::ADD;
  marker.lifetime = rclcpp::Duration::from_seconds(0);
  marker.scale.x = 0.8;
  marker.scale.y = 0.8;
  marker.scale.z = 0.8;

  for (int i = 0; i < a_cluster_vector.size(); ++i)
  {
    auto oriented_bbx = a_cluster_vector[i]->GetOrientedBoundingBox();
    geometry_msgs::msg::Point center_point;
    center_point.x = oriented_bbx.GetCenter().x();
    center_point.y = oriented_bbx.GetCenter().y();
    center_point.z = oriented_bbx.GetCenter().z();
    std_msgs::msg::ColorRGBA color;
    color.r = 1.0;
    color.a = 1.0;
    marker.colors.push_back(color);
    marker.points.push_back(center_point);
  }
  for (int i = 0; i < b_cluster_vector.size(); ++i)
  {
    auto oriented_bbx = b_cluster_vector[i]->GetOrientedBoundingBox();
    geometry_msgs::msg::Point center_point;
    center_point.x = oriented_bbx.GetCenter().x();
    center_point.y = oriented_bbx.GetCenter().y();
    center_point.z = oriented_bbx.GetCenter().z();
    std_msgs::msg::ColorRGBA color;
    color.b = 1.0;
    color.a = 1.0;
    marker.colors.push_back(color);
    marker.points.push_back(center_point);
  }
  marker_array.markers.push_back(marker);
  marker_pub_->publish(marker_array);
}

std::shared_ptr<cupoch::geometry::PointCloud> CloudSegmentation::denoiseCupochCloud(
    std::shared_ptr<cupoch::geometry::PointCloud> a,
    const std::shared_ptr<cupoch::geometry::PointCloud> b,
    double radius,
    int max_nn)
{
  auto denoised_cupoch_cloud = std::make_shared<cupoch::geometry::PointCloud>();
  cupoch::geometry::KDTreeFlann kdtree;
  kdtree.SetGeometry(*b);
  auto a_points = a->GetPoints();
  auto a_colors = a->GetColors();
  thrust::host_vector<Eigen::Vector3f> points, colors;

  int counter = 0;
  for (auto &&i : a_points)
  {
    thrust::host_vector<int> indices;
    thrust::host_vector<float> distance2;
    int k = kdtree.SearchRadius(i, radius, 100,
                                indices, distance2);
    if (k < max_nn)
    {
      points.push_back(i);
      colors.push_back(a_colors[counter]);
    }
    counter++;
  }

  denoised_cupoch_cloud->SetPoints(points);
  denoised_cupoch_cloud->SetColors(colors);
  return denoised_cupoch_cloud;
}

Eigen::Matrix4f CloudSegmentation::getTransfromfromConsecutiveOdoms(
    const nav_msgs::msg::Odometry::SharedPtr a,
    const nav_msgs::msg::Odometry::SharedPtr b)
{
  auto traveled_distance =
      Eigen::Vector3f(
          a->pose.pose.position.x - b->pose.pose.position.x,
          a->pose.pose.position.y - b->pose.pose.position.y,
          a->pose.pose.position.z - b->pose.pose.position.z)
          .norm();

  double yaw_latest, pitch_latest, roll_latest;
  double yaw, pitch, roll;

  vox_nav_utilities::getRPYfromMsgQuaternion(
      a->pose.pose.orientation, roll_latest, pitch_latest, yaw_latest);
  vox_nav_utilities::getRPYfromMsgQuaternion(
      b->pose.pose.orientation, roll, pitch, yaw);

  auto rot = cupoch::geometry::GetRotationMatrixFromXYZ(
      Eigen::Vector3f(
          roll_latest - roll, pitch_latest - pitch, yaw_latest - yaw));

  auto trans =
      Eigen::Vector3f(
          traveled_distance * cos(yaw_latest - yaw),
          traveled_distance * sin(yaw_latest - yaw), sensor_height_);

  Eigen::Matrix4f odom_T = Eigen::Matrix4f::Identity();

  odom_T.block<3, 3>(0, 0) = rot;
  odom_T.block<3, 1>(0, 3) = trans;

  return odom_T;
}

int main(int argc, char const *argv[])
{
  rclcpp::init(argc, argv);
  cupoch::utility::InitializeAllocator();
  auto node = std::make_shared<CloudSegmentation>();
  rclcpp::spin(node);
  rclcpp::shutdown();
  return 0;
}
