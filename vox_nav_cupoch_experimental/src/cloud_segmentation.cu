#include "hip/hip_runtime.h"
// Copyright (c) 2020 Fetullah Atas, Norwegian University of Life Sciences
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "vox_nav_cupoch_experimental/cloud_segmentation.hpp"

CloudSegmentation::CloudSegmentation()
        : Node("dynamic_points_node"), recieved_first_(false) {
    cloud_subscriber_.subscribe(this, "points", rmw_qos_profile_sensor_data);
    odom_subscriber_.subscribe(this, "odom", rmw_qos_profile_sensor_data);
    imu_subscriber_.subscribe(this, "imu", rmw_qos_profile_sensor_data);

    declare_parameter("dt", 0.0);
    get_parameter("dt", dt_);

    declare_parameter("sensor_height", 0.0);
    get_parameter("sensor_height", sensor_height_);

    cloud_odom_data_approx_time_syncher_.reset(new CloudOdomApprxTimeSyncer(
            CloudOdomApprxTimeSyncPolicy(500),
            cloud_subscriber_,
            odom_subscriber_,
            imu_subscriber_));

    cloud_odom_data_approx_time_syncher_->registerCallback(std::bind(
            &CloudSegmentation::cloudOdomCallback, this, std::placeholders::_1,
            std::placeholders::_2, std::placeholders::_3));

    cloud_pub_ = this->create_publisher<sensor_msgs::msg::PointCloud2>(
            "merged", rclcpp::SystemDefaultsQoS());

    marker_pub_ = this->create_publisher<visualization_msgs::msg::MarkerArray>(
            "correspondings", rclcpp::SystemDefaultsQoS());

    last_odom_msg_ = std::make_shared<nav_msgs::msg::Odometry>();
    last_dynamic_pointcloud_cupoch_ = std::make_shared<cupoch::geometry::PointCloud>();

    last_recieved_msg_stamp_ = now();
}

CloudSegmentation::~CloudSegmentation() {}

void CloudSegmentation::cloudOdomCallback(
        const sensor_msgs::msg::PointCloud2::ConstSharedPtr &cloud,
        const nav_msgs::msg::Odometry::ConstSharedPtr &odom,
        const sensor_msgs::msg::PointCloud2::ConstSharedPtr &imu) {

    if (!recieved_first_) {
        recieved_first_ = true;
        last_recieved_msg_stamp_ = cloud->header.stamp;
    }

    // convert to pcl type
    pcl::PointCloud<pcl::PointXYZRGB>::Ptr pcl_cloud(new pcl::PointCloud<pcl::PointXYZRGB>());
    pcl::fromROSMsg(*cloud, *pcl_cloud);

    sensor_msgs::PointCloud2ConstIterator<float> iter_label(*imu, "label");
    cupoch::utility::device_vector<size_t> ground_points_indices, dynamic_points_indices, static_points_indices;

    pcl::PointCloud<pcl::PointXYZRGB>::Ptr ground_points_pcl(new pcl::PointCloud<pcl::PointXYZRGB>());
    pcl::PointCloud<pcl::PointXYZRGB>::Ptr dynamic_points_pcl(new pcl::PointCloud<pcl::PointXYZRGB>());
    thrust::host_vector<Eigen::Vector3f> points, colors;

    size_t label_counter = 0;

    for (; (iter_label != iter_label.end()); ++iter_label) {
        int this_point_label = iter_label[0];
        if (this_point_label == 40) { // ground point label
            ground_points_indices.push_back(label_counter);
            ground_points_pcl->points.push_back(pcl_cloud->points[label_counter]);
        } else if (this_point_label == 30 ||
                   this_point_label == 10) { // person/car point label
            dynamic_points_indices.push_back(label_counter);
            dynamic_points_pcl->points.push_back(pcl_cloud->points[label_counter]);
        } else { // static obstacle point label
            static_points_indices.push_back(label_counter);
        }
        auto p = pcl_cloud->points[label_counter];
        points.push_back(Eigen::Vector3f(p.x, p.y, p.z));
        auto c = vox_nav_utilities::getColorByIndexEig(5);
        colors.push_back(Eigen::Vector3f(c.x(), c.y(), c.z()));
        label_counter++;
    }
    cupoch::geometry::PointCloud obstacle_cloud_cupoch;
    obstacle_cloud_cupoch.SetPoints(points);
    obstacle_cloud_cupoch.SetColors(colors);

    auto static_points_cupoch =
            obstacle_cloud_cupoch.SelectByIndex(static_points_indices, false);
    auto ground_points_cupoch =
            obstacle_cloud_cupoch.SelectByIndex(ground_points_indices, false);

    pcl::ModelCoefficients::Ptr coefficients(new pcl::ModelCoefficients);
    pcl::PointIndices::Ptr inliers(new pcl::PointIndices);
    pcl::SACSegmentation<pcl::PointXYZRGB> seg;
    seg.setOptimizeCoefficients(false);
    seg.setModelType(pcl::SACMODEL_PLANE);
    seg.setMethodType(pcl::SAC_RANSAC);
    seg.setDistanceThreshold(0.4);
    seg.setInputCloud(ground_points_pcl);
    seg.segment(*inliers, *coefficients);
    pcl::ModelOutlierRemoval<pcl::PointXYZRGB> filter;
    filter.setModelCoefficients(*coefficients);
    filter.setThreshold(0.4);
    filter.setModelType(pcl::SACMODEL_PLANE);
    filter.setInputCloud(dynamic_points_pcl);
    filter.setNegative(true);
    filter.filter(*dynamic_points_pcl);

    points.clear();
    colors.clear();
    for (int i = 0; i < dynamic_points_pcl->points.size(); ++i) {
        auto p = dynamic_points_pcl->points[i];
        points.push_back(Eigen::Vector3f(p.x, p.y, p.z));
        auto c = vox_nav_utilities::getColorByIndexEig(5);
        colors.push_back(Eigen::Vector3f(c.x(), c.y(), c.z()));
    }

    auto dynamic_points_cupoch = std::make_shared<cupoch::geometry::PointCloud>();
    dynamic_points_cupoch->SetPoints(points);
    dynamic_points_cupoch->SetColors(colors);

    dynamic_points_cupoch->PaintUniformColor(
            vox_nav_utilities::getColorByIndexEig(10));
    static_points_cupoch->PaintUniformColor(
            vox_nav_utilities::getColorByIndexEig(1));

    rclcpp::Time crr_stamp = cloud->header.stamp;
    if ((crr_stamp - last_recieved_msg_stamp_).seconds() > dt_) {

        auto travel_dist =
                Eigen::Vector3f(
                        odom->pose.pose.position.x - last_odom_msg_->pose.pose.position.x,
                        odom->pose.pose.position.y - last_odom_msg_->pose.pose.position.y,
                        odom->pose.pose.position.z - last_odom_msg_->pose.pose.position.z)
                        .norm();

        double yaw_latest, pitch_latest, roll_latest;
        double yaw, pitch, roll;

        vox_nav_utilities::getRPYfromMsgQuaternion(
                odom->pose.pose.orientation, roll_latest, pitch_latest, yaw_latest);
        vox_nav_utilities::getRPYfromMsgQuaternion(
                last_odom_msg_->pose.pose.orientation, roll, pitch, yaw);

        auto rot = cupoch::geometry::GetRotationMatrixFromXYZ(Eigen::Vector3f(
                roll_latest - roll, pitch_latest - pitch, yaw_latest - yaw));

        auto trans =
                Eigen::Vector3f(travel_dist * cos(yaw_latest - yaw),
                                travel_dist * sin(yaw_latest - yaw), sensor_height_);
        Eigen::Matrix4f odom_T = Eigen::Matrix4f::Identity();

        odom_T.block<3, 3>(0, 0) = rot;
        odom_T.block<3, 1>(0, 3) = trans;

        last_dynamic_pointcloud_cupoch_->Transform(odom_T.inverse());
        last_dynamic_pointcloud_cupoch_->PaintUniformColor(vox_nav_utilities::getColorByIndexEig(5));

        auto k =
                *dynamic_points_cupoch + *static_points_cupoch + *last_dynamic_pointcloud_cupoch_;
        auto k_ptr = std::make_shared<cupoch::geometry::PointCloud>(k);
        auto voxel_grid = cupoch::geometry::VoxelGrid::CreateFromPointCloud(k, 0.2);


        determineObjectMovements(dynamic_points_cupoch, last_dynamic_pointcloud_cupoch_, cloud->header);

        sensor_msgs::msg::PointCloud2 denoised_cloud_msg;
        cupoch_conversions::cupochToRos(k_ptr, denoised_cloud_msg,
                                        cloud->header.frame_id);
        denoised_cloud_msg.header = cloud->header;
        cloud_pub_->publish(denoised_cloud_msg);
        last_recieved_msg_stamp_ = cloud->header.stamp;
        last_odom_msg_ = std::make_shared<nav_msgs::msg::Odometry>(*odom);
        last_dynamic_pointcloud_cupoch_ =
                std::make_shared<cupoch::geometry::PointCloud>(*dynamic_points_cupoch);
    }
}

void CloudSegmentation::determineObjectMovements(std::shared_ptr<cupoch::geometry::PointCloud> a,
                                                 std::shared_ptr<cupoch::geometry::PointCloud> b,
                                                 std_msgs::msg::Header header) {

    if (!a->points_.size() || !b->points_.size()) {
        RCLCPP_INFO(get_logger(),
                    "Passing this cycle of object movemnet as one of the cloud is empty, clouds have a: %d b: %d points",
                    a->points_.size(), b->points_.size());
        return;
    }

    // REMOVE THE NOISE
    auto denoised_a = a->RemoveStatisticalOutliers(10, 0.1);
    auto denoised_b = b->RemoveStatisticalOutliers(10, 0.1);
    denoised_a = std::get<0>(denoised_a)->RemoveRadiusOutliers(2, 0.2);
    denoised_b = std::get<0>(denoised_b)->RemoveRadiusOutliers(2, 0.2);
    a = std::get<0>(denoised_a);
    b = std::get<0>(denoised_b);

    auto knn_search = cupoch::geometry::KDTreeSearchParamKNN(10);
    a->EstimateNormals(knn_search);
    b->EstimateNormals(knn_search);

    auto a_feature = cupoch::registration::ComputeFPFHFeature(*a, knn_search);
    auto b_feature = cupoch::registration::ComputeFPFHFeature(*b, knn_search);

    cupoch::registration::FastGlobalRegistrationOption regist_option;
    regist_option.maximum_correspondence_distance_ = 0.8;

    cupoch::registration::RegistrationResult regist_result = cupoch::registration::FastGlobalRegistration(*a, *b,
                                                                                                          *a_feature,
                                                                                                          *b_feature,
                                                                                                          regist_option);
    thrust::host_vector<Eigen::Matrix<int, 2, 1>> corresponding_set = regist_result.GetCorrespondenceSet();

    visualization_msgs::msg::MarkerArray marker_array;
    visualization_msgs::msg::Marker marker;
    marker.header = header;
    marker.type = visualization_msgs::msg::Marker::LINE_LIST;
    marker.action = visualization_msgs::msg::Marker::ADD;
    marker.lifetime = rclcpp::Duration::from_seconds(0);
    marker.scale.x = 0.8;
    marker.scale.y = 0.2;
    marker.scale.z = 0.2;
    auto a_points = a->GetPoints();
    auto b_points = b->GetPoints();

    for (size_t i = 0; i < corresponding_set.size(); ++i) {
        auto curr_set = corresponding_set[i];

        geometry_msgs::msg::Point first_point, second_point;
        first_point.x = a_points[curr_set(0, 0)].x();
        first_point.y = a_points[curr_set(0, 0)].y();
        first_point.z = a_points[curr_set(0, 0)].z();
        second_point.x = b_points[curr_set(1, 0)].x();
        second_point.y = b_points[curr_set(1, 0)].y();
        second_point.z = b_points[curr_set(1, 0)].z();

        marker.points.push_back(first_point);
        marker.points.push_back(second_point);

        std_msgs::msg::ColorRGBA color;
        color.r = 1.0;
        color.a = 1.0;

        marker.colors.push_back(color);
        marker.colors.push_back(color);
    }

    marker_array.markers.push_back(marker);

    marker_pub_->publish(marker_array);

}

std::vector<geometry_msgs::msg::Point>
CloudSegmentation::Vector3List2GeometryMsgs(
        ApproxMVBB::TypeDefsPoints::Vector3List corners) {
    std::vector<geometry_msgs::msg::Point> corners_geometry_msgs;
    for (int i = 0; i < corners.size(); i++) {
        geometry_msgs::msg::Point korner_point;
        korner_point.x = corners[i].x();
        korner_point.y = corners[i].y();
        korner_point.z = corners[i].z();
        corners_geometry_msgs.push_back(korner_point);
    }
    return corners_geometry_msgs;
}

std::vector<geometry_msgs::msg::Point> CloudSegmentation::Eigen2GeometryMsgs(
        std::array<Eigen::Matrix<float, 3, 1>, 8> obbx_corners) {
    std::vector<geometry_msgs::msg::Point> corners_geometry_msgs;
    for (int i = 0; i < obbx_corners.size(); i++) {
        geometry_msgs::msg::Point korner_point;
        korner_point.x = obbx_corners[i].x();
        korner_point.y = obbx_corners[i].y();
        korner_point.z = obbx_corners[i].z();
        corners_geometry_msgs.push_back(korner_point);
    }
    return corners_geometry_msgs;
}

cupoch::utility::device_vector<Eigen::Vector3f>
CloudSegmentation::Vector3List2Eigen(
        ApproxMVBB::TypeDefsPoints::Vector3List corners) {
    cupoch::utility::device_vector<Eigen::Vector3f> corners_eigen;
    for (int i = 0; i < corners.size(); i++) {
        Eigen::Vector3f korner_point;
        korner_point.x() = corners[i].x();
        korner_point.y() = corners[i].y();
        korner_point.z() = corners[i].z();
        corners_eigen.push_back(korner_point);
    }
    return corners_eigen;
}

int main(int argc, char const *argv[]) {
    rclcpp::init(argc, argv);
    cupoch::utility::InitializeAllocator();
    auto node = std::make_shared<CloudSegmentation>();
    rclcpp::spin(node);
    rclcpp::shutdown();
    return 0;
}
