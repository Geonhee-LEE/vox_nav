#include "hip/hip_runtime.h"
// Copyright (c) 2020 Fetullah Atas, Norwegian University of Life Sciences
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "message_filters/subscriber.h"
#include "message_filters/sync_policies/approximate_time.h"
#include "message_filters/synchronizer.h"
#include "tf2_geometry_msgs/tf2_geometry_msgs.h"
#include <nav_msgs/msg/odometry.hpp>
#include <rclcpp/rclcpp.hpp>
#include <sensor_msgs/msg/imu.hpp>
#include <sensor_msgs/msg/point_cloud2.hpp>
#include <visualization_msgs/msg/marker_array.hpp>

#include "pcl/filters/model_outlier_removal.h"
#include <pcl/PCLPointCloud2.h>
#include <pcl/common/common.h>
#include <pcl/conversions.h>
#include <pcl/filters/crop_box.h>
#include <pcl/filters/extract_indices.h>
#include <pcl/io/pcd_io.h>
#include <pcl/point_cloud.h>
#include <pcl/point_types.h>
#include <pcl_conversions/pcl_conversions.h>
#include <pcl_ros/transforms.hpp>
#include <tf2_eigen/tf2_eigen.h>

#include <boost/geometry.hpp>
#include <fstream>
#include <iomanip>
#include <iostream>
#include <ostream>
#include <queue>
#include <sstream>

#include "ApproxMVBB/AABB.hpp"
#include "ApproxMVBB/ComputeApproxMVBB.hpp"
#include "cupoch/collision/collision.h"
#include "cupoch/cupoch.h"
#include "cupoch/geometry/occupancygrid.h"
#include "cupoch_conversions/cupoch_conversions.hpp"
#include "std_msgs/msg/u_int8_multi_array.hpp"
#include "vox_nav_cupoch_experimental/visibility_control.h"
#include "vox_nav_utilities/map_manager_helpers.hpp"
#include "vox_nav_utilities/pcl_helpers.hpp"
#include "vox_nav_utilities/tf_helpers.hpp"

namespace bg = boost::geometry;

class CloudSegmentation : public rclcpp::Node {

public:
  typedef std::tuple<sensor_msgs::msg::PointCloud2::SharedPtr,
                     nav_msgs::msg::Odometry::SharedPtr,
                     sensor_msgs::msg::PointCloud2::SharedPtr>
      data_captute_t;

  CloudSegmentation();
  ~CloudSegmentation();

  typedef message_filters::sync_policies::ApproximateTime<
      sensor_msgs::msg::PointCloud2, nav_msgs::msg::Odometry,
      sensor_msgs::msg::PointCloud2>
      CloudOdomApprxTimeSyncPolicy;
  typedef message_filters::Synchronizer<CloudOdomApprxTimeSyncPolicy>
      CloudOdomApprxTimeSyncer;

  void
  cloudOdomCallback(const sensor_msgs::msg::PointCloud2::ConstSharedPtr &cloud,
                    const nav_msgs::msg::Odometry::ConstSharedPtr &odom,
                    const sensor_msgs::msg::PointCloud2::ConstSharedPtr &imu);

  pcl::PointCloud<pcl::PointXYZRGB>::Ptr
  denoise_segmented_cloud(const pcl::PointCloud<pcl::PointXYZRGB>::Ptr cloud,
                          double radius, double tolerated_divergence_rate,
                          int min_num_neighbours);

  void shoot(std::vector<data_captute_t> &cloud_vector);

  std::vector<geometry_msgs::msg::Point>
  Vector3List2GeometryMsgs(ApproxMVBB::TypeDefsPoints::Vector3List corners);

  std::vector<geometry_msgs::msg::Point>
  Eigen2GeometryMsgs(std::array<Eigen::Matrix<float, 3, 1>, 8> obbx_corners);

  cupoch::utility::device_vector<Eigen::Vector3f>
  Vector3List2Eigen(ApproxMVBB::TypeDefsPoints::Vector3List corners);

private:
  rclcpp::TimerBase::SharedPtr timer_;
  message_filters::Subscriber<sensor_msgs::msg::PointCloud2> cloud_subscriber_;
  message_filters::Subscriber<nav_msgs::msg::Odometry> odom_subscriber_;
  message_filters::Subscriber<sensor_msgs::msg::PointCloud2> imu_subscriber_;
  std::shared_ptr<CloudOdomApprxTimeSyncer>
      cloud_odom_data_approx_time_syncher_;

  double dt_;
  double sensor_height_;

  std::vector<data_captute_t> cloud_odom_vector_;

  bool recieved_first_;

  rclcpp::Publisher<sensor_msgs::msg::PointCloud2>::SharedPtr pub_;
  rclcpp::Publisher<sensor_msgs::msg::PointCloud2>::SharedPtr dyn_point_pub_;
  rclcpp::Publisher<visualization_msgs::msg::MarkerArray>::SharedPtr vis_pub_;

  rclcpp::Time last_recieved_msg_stamp_;

  std::shared_ptr<cupoch::geometry::PointCloud> last_pointcloud_cupoch_;
  nav_msgs::msg::Odometry::SharedPtr last_odom_msg_;
};

CloudSegmentation::CloudSegmentation()
    : Node("dynamic_points_node"), recieved_first_(false) {
  cloud_subscriber_.subscribe(this, "points", rmw_qos_profile_sensor_data);
  odom_subscriber_.subscribe(this, "odom", rmw_qos_profile_sensor_data);
  imu_subscriber_.subscribe(this, "imu", rmw_qos_profile_sensor_data);

  declare_parameter("dt", 0.0);
  get_parameter("dt", dt_);

  declare_parameter("sensor_height", 0.0);
  get_parameter("sensor_height", sensor_height_);

  cloud_odom_data_approx_time_syncher_.reset(new CloudOdomApprxTimeSyncer(
      CloudOdomApprxTimeSyncPolicy(1000), cloud_subscriber_, odom_subscriber_,
      imu_subscriber_));

  cloud_odom_data_approx_time_syncher_->registerCallback(std::bind(
      &CloudSegmentation::cloudOdomCallback, this, std::placeholders::_1,
      std::placeholders::_2, std::placeholders::_3));

  pub_ = this->create_publisher<sensor_msgs::msg::PointCloud2>(
      "merged", rclcpp::SystemDefaultsQoS());

  dyn_point_pub_ = this->create_publisher<sensor_msgs::msg::PointCloud2>(
      "dyn_point_pub", rclcpp::SystemDefaultsQoS());

  vis_pub_ = this->create_publisher<visualization_msgs::msg::MarkerArray>(
      "markers", rclcpp::SystemDefaultsQoS());

  auto captured_data = std::make_tuple<>(
      std::make_shared<sensor_msgs::msg::PointCloud2>(
          sensor_msgs::msg::PointCloud2()),
      std::make_shared<nav_msgs::msg::Odometry>(nav_msgs::msg::Odometry()),
      std::make_shared<sensor_msgs::msg::PointCloud2>(
          sensor_msgs::msg::PointCloud2()));

  cloud_odom_vector_ = std::vector<data_captute_t>(2, captured_data);

  last_odom_msg_ = std::make_shared<nav_msgs::msg::Odometry>();
  last_pointcloud_cupoch_ = std::make_shared<cupoch::geometry::PointCloud>();

  last_recieved_msg_stamp_ = now();
}

CloudSegmentation::~CloudSegmentation() {}

void CloudSegmentation::cloudOdomCallback(
    const sensor_msgs::msg::PointCloud2::ConstSharedPtr &cloud,
    const nav_msgs::msg::Odometry::ConstSharedPtr &odom,
    const sensor_msgs::msg::PointCloud2::ConstSharedPtr &imu) {

  if (!recieved_first_) {
    recieved_first_ = true;
    last_recieved_msg_stamp_ = cloud->header.stamp;
  }

  // convert to pcl type
  pcl::PointCloud<pcl::PointXYZRGB>::Ptr curr_pcl(
      new pcl::PointCloud<pcl::PointXYZRGB>());
  pcl::PCLPointCloud2 pcl_pc;
  pcl_conversions::toPCL(*cloud, pcl_pc);
  pcl::fromPCLPointCloud2(pcl_pc, *curr_pcl);

  sensor_msgs::PointCloud2ConstIterator<float> iter_label(*imu, "label");

  cupoch::utility::device_vector<size_t> ground_points_indices;
  cupoch::utility::device_vector<size_t> dynamic_points_indices;
  cupoch::utility::device_vector<size_t> static_points_indices;

  size_t label_counter = 0;
  for (; (iter_label != iter_label.end()); ++iter_label) {
    curr_pcl->points[label_counter].r = iter_label[0];
    label_counter++;
  }

  /*curr_pcl = vox_nav_utilities::removeOutliersFromInputCloud(
    curr_pcl, 4, 0.5,
    vox_nav_utilities::OutlierRemovalType::RadiusOutlierRemoval);
  curr_pcl =
      vox_nav_utilities::downsampleInputCloud<pcl::PointXYZRGB>(curr_pcl,
  0.05);*/

  cupoch::geometry::PointCloud obstacle_cloud_cupoch;
  thrust::host_vector<Eigen::Vector3f> points;
  thrust::host_vector<Eigen::Vector3f> colors;

  for (int i = 0; i < curr_pcl->points.size(); ++i) {
    int this_point_label = curr_pcl->points[i].r;
    if (this_point_label == 40) { // ground point label
      ground_points_indices.push_back(i);
    } else if (this_point_label == 30 ||
               this_point_label == 10) { // person/car point label
      dynamic_points_indices.push_back(i);
    } else { // static obstacle point label
      static_points_indices.push_back(i);
    }
  }

  for (int i = 0; i < curr_pcl->points.size(); ++i) {
    auto p = curr_pcl->points[i];
    points.push_back(Eigen::Vector3f(p.x, p.y, p.z));
    auto c = vox_nav_utilities::getColorByIndexEig(5);
    colors.push_back(Eigen::Vector3f(c.x(), c.y(), c.z()));
  }
  obstacle_cloud_cupoch.SetPoints(points);
  obstacle_cloud_cupoch.SetColors(colors);

  auto dynamic_points_cupoch =
      obstacle_cloud_cupoch.SelectByIndex(dynamic_points_indices, false);

  auto static_points_cupoch =
      obstacle_cloud_cupoch.SelectByIndex(static_points_indices, false);

  auto ground_points_cupoch =
      obstacle_cloud_cupoch.SelectByIndex(ground_points_indices, false);

  pcl::PointCloud<pcl::PointXYZRGB>::Ptr ground_points_pcl(
      new pcl::PointCloud<pcl::PointXYZRGB>());

  pcl::PointCloud<pcl::PointXYZRGB>::Ptr dynamic_points_pcl(
      new pcl::PointCloud<pcl::PointXYZRGB>());

  for (auto &&h : ground_points_cupoch->points_) {
    pcl::PointXYZRGB point;
    Eigen::Matrix<float, 3, 1> g = h;
    point.x = g.x();
    point.y = g.y();
    point.z = g.z();
    ground_points_pcl->points.push_back(point);
  }

  for (auto &&h : dynamic_points_cupoch->points_) {
    pcl::PointXYZRGB point;
    Eigen::Matrix<float, 3, 1> g = h;
    point.x = g.x();
    point.y = g.y();
    point.z = g.z();
    dynamic_points_pcl->points.push_back(point);
  }

  pcl::ModelCoefficients::Ptr coefficients(new pcl::ModelCoefficients);
  pcl::PointIndices::Ptr inliers(new pcl::PointIndices);
  pcl::SACSegmentation<pcl::PointXYZRGB> seg;
  seg.setOptimizeCoefficients(true);     // Optional
  seg.setModelType(pcl::SACMODEL_PLANE); // PLANE
  seg.setMethodType(pcl::SAC_RANSAC);    // RANSAC
  seg.setDistanceThreshold(0.2); // determines how close a point must be to the
                                 // model in order to be considered an inlier
  seg.setInputCloud(ground_points_pcl);
  seg.segment(*inliers, *coefficients);

  pcl::ModelOutlierRemoval<pcl::PointXYZRGB> filter;
  filter.setModelCoefficients(*coefficients);
  filter.setThreshold(0.2);
  filter.setModelType(pcl::SACMODEL_PLANE);
  filter.setInputCloud(dynamic_points_pcl);
  filter.setNegative(true);
  filter.filter(*dynamic_points_pcl);

  points.clear();
  colors.clear();

  for (int i = 0; i < dynamic_points_pcl->points.size(); ++i) {
    auto p = dynamic_points_pcl->points[i];
    points.push_back(Eigen::Vector3f(p.x, p.y, p.z));
    auto c = vox_nav_utilities::getColorByIndexEig(5);
    colors.push_back(Eigen::Vector3f(c.x(), c.y(), c.z()));
  }
  dynamic_points_cupoch->SetPoints(points);
  dynamic_points_cupoch->SetColors(colors);

  dynamic_points_cupoch->PaintUniformColor(
      vox_nav_utilities::getColorByIndexEig(2));

  static_points_cupoch->PaintUniformColor(
      vox_nav_utilities::getColorByIndexEig(1));

  rclcpp::Time crr_stamp = cloud->header.stamp;
  if ((crr_stamp - last_recieved_msg_stamp_).seconds() > dt_) {
    auto travel_dist =
        Eigen::Vector3f(
            odom->pose.pose.position.x - last_odom_msg_->pose.pose.position.x,
            odom->pose.pose.position.y - last_odom_msg_->pose.pose.position.y,
            odom->pose.pose.position.z - last_odom_msg_->pose.pose.position.z)
            .norm();

    double yaw_latest, pitch_latest, roll_latest;
    double yaw, pitch, roll;

    vox_nav_utilities::getRPYfromMsgQuaternion(
        odom->pose.pose.orientation, roll_latest, pitch_latest, yaw_latest);
    vox_nav_utilities::getRPYfromMsgQuaternion(
        last_odom_msg_->pose.pose.orientation, roll, pitch, yaw);

    auto rot = cupoch::geometry::GetRotationMatrixFromXYZ(Eigen::Vector3f(
        roll_latest - roll, pitch_latest - pitch, yaw_latest - yaw));

    auto trans =
        Eigen::Vector3f(travel_dist * cos(yaw_latest - yaw),
                        travel_dist * sin(yaw_latest - yaw), sensor_height_);
    Eigen::Matrix4f odom_T = Eigen::Matrix4f::Identity();

    odom_T.block<3, 3>(0, 0) = rot;
    odom_T.block<3, 1>(0, 3) = trans;

    auto last_pointcloud_cupoch =
        last_pointcloud_cupoch_->Transform(odom_T.inverse());
    last_pointcloud_cupoch = last_pointcloud_cupoch_->PaintUniformColor(
        vox_nav_utilities::getColorByIndexEig(5));

    auto k =
        *dynamic_points_cupoch + *static_points_cupoch + last_pointcloud_cupoch;
    auto k_ptr = std::make_shared<cupoch::geometry::PointCloud>(k);
    auto voxel_grid = cupoch::geometry::VoxelGrid::CreateFromPointCloud(k, 0.2);

    sensor_msgs::msg::PointCloud2 denoised_cloud_msg;
    cupoch_conversions::cupochToRos(k_ptr, denoised_cloud_msg,
                                    cloud->header.frame_id);
    denoised_cloud_msg.header = cloud->header;
    pub_->publish(denoised_cloud_msg);

    last_recieved_msg_stamp_ = cloud->header.stamp;
    last_odom_msg_ = std::make_shared<nav_msgs::msg::Odometry>(*odom);
    last_pointcloud_cupoch_ =
        std::make_shared<cupoch::geometry::PointCloud>(*dynamic_points_cupoch);
  }

  /*
    auto occupancy_grid =
      cupoch::geometry::OccupancyGrid::CreateFromVoxelGrid(*voxel_grid);
    auto distance_transform =
      cupoch::geometry::DistanceTransform::CreateFromOccupancyGrid(
          *occupancy_grid);*/
}

void CloudSegmentation::shoot(std::vector<data_captute_t> &cloud_vector) {}

std::vector<geometry_msgs::msg::Point>
CloudSegmentation::Vector3List2GeometryMsgs(
    ApproxMVBB::TypeDefsPoints::Vector3List corners) {
  std::vector<geometry_msgs::msg::Point> corners_geometry_msgs;
  for (int i = 0; i < corners.size(); i++) {
    geometry_msgs::msg::Point korner_point;
    korner_point.x = corners[i].x();
    korner_point.y = corners[i].y();
    korner_point.z = corners[i].z();
    corners_geometry_msgs.push_back(korner_point);
  }
  return corners_geometry_msgs;
}

std::vector<geometry_msgs::msg::Point> CloudSegmentation::Eigen2GeometryMsgs(
    std::array<Eigen::Matrix<float, 3, 1>, 8> obbx_corners) {
  std::vector<geometry_msgs::msg::Point> corners_geometry_msgs;
  for (int i = 0; i < obbx_corners.size(); i++) {
    geometry_msgs::msg::Point korner_point;
    korner_point.x = obbx_corners[i].x();
    korner_point.y = obbx_corners[i].y();
    korner_point.z = obbx_corners[i].z();
    corners_geometry_msgs.push_back(korner_point);
  }
  return corners_geometry_msgs;
}

cupoch::utility::device_vector<Eigen::Vector3f>
CloudSegmentation::Vector3List2Eigen(
    ApproxMVBB::TypeDefsPoints::Vector3List corners) {
  cupoch::utility::device_vector<Eigen::Vector3f> corners_eigen;
  for (int i = 0; i < corners.size(); i++) {
    Eigen::Vector3f korner_point;
    korner_point.x() = corners[i].x();
    korner_point.y() = corners[i].y();
    korner_point.z() = corners[i].z();
    corners_eigen.push_back(korner_point);
  }
  return corners_eigen;
}

int main(int argc, char const *argv[]) {
  rclcpp::init(argc, argv);
  cupoch::utility::InitializeAllocator();
  auto node = std::make_shared<CloudSegmentation>();
  rclcpp::spin(node);
  rclcpp::shutdown();
  return 0;
}
