#include "hip/hip_runtime.h"
#include "vox_nav_cupoch_experimental/clustering.hpp"

Clustering::Clustering()
        : Node("clustering_node") {
    cloud_subscriber_.subscribe(this, "points", rmw_qos_profile_sensor_data);
    odom_subscriber_.subscribe(this, "odom", rmw_qos_profile_sensor_data);

    cloud_odom_data_approx_time_syncher_.reset(
            new CloudOdomApprxTimeSyncer(
                    CloudOdomApprxTimeSyncPolicy(500),
                    cloud_subscriber_,
                    odom_subscriber_));

    cloud_odom_data_approx_time_syncher_->registerCallback(
            std::bind(
                    &Clustering::cloudOdomCallback, this,
                    std::placeholders::_1,
                    std::placeholders::_2));

    cloud_pub_ = this->create_publisher<sensor_msgs::msg::PointCloud2>(
            "merged", rclcpp::SystemDefaultsQoS());

    marker_pub_ = this->create_publisher<visualization_msgs::msg::MarkerArray>(
            "correspondings", rclcpp::SystemDefaultsQoS());

    last_recieved_msg_stamp_ = now();

}

Clustering::~Clustering() {
}

void Clustering::cloudOdomCallback(
        const sensor_msgs::msg::PointCloud2::ConstSharedPtr &cloud,
        const nav_msgs::msg::Odometry::ConstSharedPtr &odom) {
    pcl::PointCloud<pcl::PointXYZRGB>::Ptr pcl_curr(new pcl::PointCloud<pcl::PointXYZRGB>());
    pcl::fromROSMsg(*cloud, *pcl_curr);

    pcl_curr = vox_nav_utilities::crop_box<pcl::PointXYZRGB>(
            pcl_curr, Eigen::Vector4f(-15, -15, -2, 1), Eigen::Vector4f(15, 15, 2, 1));
    pcl_curr = vox_nav_utilities::downsampleInputCloud<pcl::PointXYZRGB>(pcl_curr, 0.05);
    pcl_curr = vox_nav_utilities::segmentSurfacePlane<pcl::PointXYZRGB>(pcl_curr, 0.4, true);
    auto clusters = vox_nav_utilities::euclidean_clustering<pcl::PointXYZRGB>(
            pcl_curr, 20, 10000, 0.32);
    vox_nav_utilities::publishClustersCloud(cloud_pub_, cloud->header, clusters);

    std::vector<std::pair<Eigen::Vector3f, Eigen::Vector3f>> boxes_vector;
    for (auto &&cluster: clusters) {
        if (cluster->points.size() < 10) {
            RCLCPP_WARN(this->get_logger(), "THIS OBJECT HAVE TOO FEW POINTS NOT GONNA BUILD A BOX !!");
            continue;
        }
        auto cupoch_cloud = std::make_shared<cupoch::geometry::PointCloud>();
        thrust::host_vector<Eigen::Vector3f> points;
        for (auto &&i: cluster->points) {
            points.push_back(Eigen::Vector3f(i.x, i.y, i.z));
        }
        cupoch_cloud->SetPoints(points);
        auto oobb = cupoch_cloud->GetAxisAlignedBoundingBox();
        boxes_vector.push_back(std::make_pair(oobb.GetMinBound(), oobb.GetMaxBound()));

    }

    visualization_msgs::msg::MarkerArray marker_array;
    for (size_t i = 0; i < boxes_vector.size(); i++) {
        auto mvbb_corners_geometry_msgs = boxes_vector[i];
        visualization_msgs::msg::Marker marker;
        marker.header = cloud->header;
        marker.ns = "my_namespace";
        marker.id = i;
        marker.lifetime = rclcpp::Duration::from_seconds(1.0);
        marker.type = visualization_msgs::msg::Marker::LINE_LIST;
        marker.action = visualization_msgs::msg::Marker::ADD;
        marker.scale.x = 0.1;
        marker.scale.y = 0.1;
        marker.scale.z = 0.1;
        marker.color.a = 1.0;
        marker.color.r = 1.0;
        marker.color.g = 1.0;
        marker.color.b = 0.0;

        geometry_msgs::msg::Point p1;
        p1.x = mvbb_corners_geometry_msgs.first.x();
        p1.y = mvbb_corners_geometry_msgs.first.y();
        p1.z = mvbb_corners_geometry_msgs.first.z();

        marker.points.push_back(p1);
        /*marker.points.push_back(mvbb_corners_geometry_msgs[1]);
        marker.points.push_back(mvbb_corners_geometry_msgs[2]);
        marker.points.push_back(mvbb_corners_geometry_msgs[3]);
        marker.points.push_back(mvbb_corners_geometry_msgs[0]);
        marker.points.push_back(mvbb_corners_geometry_msgs[4]);
        marker.points.push_back(mvbb_corners_geometry_msgs[5]);
        marker.points.push_back(mvbb_corners_geometry_msgs[6]);
        marker.points.push_back(mvbb_corners_geometry_msgs[7]);
        marker.points.push_back(mvbb_corners_geometry_msgs[4]);
        marker.points.push_back(mvbb_corners_geometry_msgs[0]);
        marker.points.push_back(mvbb_corners_geometry_msgs[4]);
        marker.points.push_back(mvbb_corners_geometry_msgs[1]);
        marker.points.push_back(mvbb_corners_geometry_msgs[5]);
        marker.points.push_back(mvbb_corners_geometry_msgs[2]);
        marker.points.push_back(mvbb_corners_geometry_msgs[3]);
        marker.points.push_back(mvbb_corners_geometry_msgs[6]);*/

        geometry_msgs::msg::Point p2;
        p2.x = mvbb_corners_geometry_msgs.second.x();
        p2.y = mvbb_corners_geometry_msgs.second.y();
        p2.z = mvbb_corners_geometry_msgs.second.z();
        marker.points.push_back(p2);
        marker_array.markers.push_back(marker);

        visualization_msgs::msg::Marker cy_marker;
        cy_marker.header = cloud->header;
        cy_marker.ns = "my_namespace";
        cy_marker.id = i + 50;
        cy_marker.lifetime = rclcpp::Duration::from_seconds(1.0);
        cy_marker.type = visualization_msgs::msg::Marker::CYLINDER;
        cy_marker.action = visualization_msgs::msg::Marker::ADD;
        double scale = 1.25;
        cy_marker.scale.x =
                scale * std::abs(mvbb_corners_geometry_msgs.second.x() - mvbb_corners_geometry_msgs.first.x());
        cy_marker.scale.y =
                scale * std::abs(mvbb_corners_geometry_msgs.second.y() - mvbb_corners_geometry_msgs.first.y());
        cy_marker.scale.z =
                scale * std::abs(mvbb_corners_geometry_msgs.second.z() - mvbb_corners_geometry_msgs.first.z());
        cy_marker.color.a = 1.0;
        cy_marker.color.r = 1.0;
        cy_marker.color.g = 0.6;
        cy_marker.color.b = 0.6;
        auto cy_rotation = std::atan2(
                (mvbb_corners_geometry_msgs.second.x() - mvbb_corners_geometry_msgs.second.x()),
                (mvbb_corners_geometry_msgs.second.y() - mvbb_corners_geometry_msgs.first.y())
        );
        double r, p, y;
        //cy_marker.pose.orientation = vox_nav_utilities::getMsgQuaternionfromRPY(r, p, cy_rotation);
        cy_marker.pose.position.x =
                (mvbb_corners_geometry_msgs.second.x() + mvbb_corners_geometry_msgs.first.x()) / 2.0;
        cy_marker.pose.position.y =
                (mvbb_corners_geometry_msgs.second.y() + mvbb_corners_geometry_msgs.first.y()) / 2.0;
        cy_marker.pose.position.z =
                (mvbb_corners_geometry_msgs.second.z() + mvbb_corners_geometry_msgs.first.z()) / 2.0;
        marker_array.markers.push_back(cy_marker);
    }
    marker_pub_->publish(marker_array);
}

std::vector<geometry_msgs::msg::Point> Clustering::Vector3List2GeometryMsgs(
        ApproxMVBB::TypeDefsPoints::Vector3List corners) {
    std::vector<geometry_msgs::msg::Point> corners_geometry_msgs;
    for (int i = 0; i < corners.size(); i++) {
        geometry_msgs::msg::Point korner_point;
        korner_point.x = corners[i].x();
        korner_point.y = corners[i].y();
        korner_point.z = corners[i].z();
        corners_geometry_msgs.push_back(korner_point);
    }
    return corners_geometry_msgs;
}

int main(int argc, char const *argv[]) {
    rclcpp::init(argc, argv);
    auto node = std::make_shared<Clustering>();
    rclcpp::spin(node);
    rclcpp::shutdown();
    return 0;
}
